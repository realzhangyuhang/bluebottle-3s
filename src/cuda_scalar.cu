#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>

#include "cuda_particle.h"
#include "scalar.h"
#include "cuda_scalar.h"

#include <hip/hip_runtime_api.h>
//#include <thrust/scan.h>
//#include <thrust/device_ptr.h>

__constant__ int _s_mm[64];
__constant__ int _s_nn[64];
real *_int_Ys_re;
real *_int_Ys_im;

extern "C"
void cuda_scalar_transfer_parts_i(void)
{
  //printf("N%d >> Transfering parts in i, nparts = %d\n", rank, nparts);
  /* Transfer particles east and west
   *  * Bin the particles, indexing with `i` varying slowest
   *  * Sort particles by their bin
   *  * Find start and end of each bin's particles
   *  * Find number of particles in each bin
   *  * Find number of particles in _is & _ie planes. These need to be sent W/E
   *  * Communicate these number east and west. Each process now knows how many
   *    to send and recv
   *  * Allocate memory for particle send and recv
   *  * Copy particles into sending arrays. Each bin can find the offset target
   *    index for its particles by performing a prefix scan.
   *  * Communicate particles east and west, send -> recv
   *  * Recv'd parts exist in the ghost bins and replace whatever existed there
   *    at the last time step. Sum the particles in _isb & _ieb and subtract
   *    from nparts. This, plus the number of particle recv'd from E/W, is the
   *    number of new particles
   *  * Allocate temp part structure to hold all new particles.
   *  * Reduce bin_count from _is->_ie to find nparts that we will keep
   *  * Prefix scan from _ie -> _ie to find offset index for particle copy to
   *    temp struct
   *  * Backfill recv'd particles to the end of the temp array
   *  * Repeat process for j, k to take care of edge, corner. Indexing will be
   *    different to take advantage of memory coalescence and the prefix scan
   *    two steps back
   */

  /* NOTE
   *  cuda-memcheck occasionally produces the error "bulk_kernel_by_value: an
   *  illegal memory address was encountered" error on a (thrust) call to
   *  hipDeviceSynchronize. This doesn't seem to be reliably reproducible
   *  (occurs on any of the several thrust calls in this function). This does
   *  not seem to affect the results in any way, but should be further
   *  investigated. See bug id 008.
   */

  /* Init execution config -- thread over east/west faces */
  int ty = bins.Gcc.jnb * (bins.Gcc.jnb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.jnb >= MAX_THREADS_DIM);
  int tz = bins.Gcc.knb * (bins.Gcc.knb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.knb >= MAX_THREADS_DIM);

  int by = (int) ceil((real) bins.Gcc.jnb / (real) ty);
  int bz = (int) ceil((real) bins.Gcc.knb / (real) tz);

  dim3 bin_num_inb(by, bz);
  dim3 bin_dim_inb(ty, tz);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b, s2b; // custom strides
  int offset;

  /* Allocate memory */
  // These are realloc'd every time
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_e;
  int *_offset_w;
  checkCudaErrors(hipMalloc(&_offset_e, bins.Gcc.s2b_i * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_w, bins.Gcc.s2b_i * sizeof(int)));
  thrust::device_ptr<int> t_offset_e(_offset_e);
  thrust::device_ptr<int> t_offset_w(_offset_w);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  // If we have parts...
  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_i<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_i<<<bin_num_inb, bin_dim_inb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send, and packing offsets */
    s1b = bins.Gcc.jnb;
    s2b = s1b * bins.Gcc.knb;

    // East
    offset = GFX_LOC(bins.Gcc._ie, 0, 0, s1b, s2b);
    if (dom[rank].e != MPI_PROC_NULL) {
      // _bin_count is indexed with i varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _ie plane
      nparts_send[EAST] = thrust::reduce(t_bin_count + offset,
                                         t_bin_count + offset + bins.Gcc.s2b_i,
                                         0., thrust::plus<int>());

      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[EAST] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_i, t_offset_e);
      } else {
        hipMemset(_offset_e, 0., bins.Gcc.s2b_i * sizeof(int));
      }

    } else { // no parts to send
      nparts_send[EAST] = 0;
      hipMemset(_offset_e, 0., bins.Gcc.s2b_i * sizeof(int));
    }

    // West
    offset = GFX_LOC(bins.Gcc._is, 0, 0, s1b, s2b);
    if (dom[rank].w != MPI_PROC_NULL) {
      nparts_send[WEST] = thrust::reduce(t_bin_count + offset,
                                         t_bin_count + offset + bins.Gcc.s2b_i,
                                         0., thrust::plus<int>());
      if (nparts_send[WEST] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_i, t_offset_w);
      } else {
        hipMemset(_offset_w, 0., bins.Gcc.s2b_i * sizeof(int));
      }

    } else {
      nparts_send[WEST] = 0;
      hipMemset(_offset_w, 0., bins.Gcc.s2b_i * sizeof(int));
    }

  } else { // nparts <= 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[EAST] = 0;
    nparts_send[WEST] = 0;
    hipMemset(_offset_e, 0., bins.Gcc.s2b_i * sizeof(int));
    hipMemset(_offset_w, 0., bins.Gcc.s2b_i * sizeof(int));
  }

  /* Send number of parts to east/west */
  //    origin                target
  // nparts_send[WEST] -> nparts_recv[EAST]
  // nparts_recv[WEST] <- nparts_send[EAST]
  nparts_recv[WEST] = 0; // init
  nparts_recv[EAST] = 0;
  mpi_send_nparts_i();

  /* Allocate memory for send and receiving particles */
  // NOTE: If no particles need to be sent/received in a given direction, this
  //  allocates a memory location with size zero which returns a null device
  //  pointer. If this is passed to MPI_Win_create(base, ...) as the base in
  //  CUDA 9.0, it causes MPI to hang. This was not an issue in CUDA 7.5
  //
  // The fix involves fooling MPI by allocating a very small amount of dummy
  // information if no particles are to be sent. This gives the location a valid
  // memory pointer, than than a null pointer. The MPI communication still knows
  // that the allocated window size and info to be sent is zero, and nothing is
  // unpacked because that is wrapped in an if-statement already. This doesn't 
  // affect most cases where particles are communicated every direction at every
  // time; this will only affect extremely dilute cases.

  int send_alloc_e = nparts_send[EAST]*(nparts_send[EAST] > 0) + (nparts_send[EAST] == 0);
  int send_alloc_w = nparts_send[WEST]*(nparts_send[WEST] > 0) + (nparts_send[WEST] == 0);
  int recv_alloc_e = nparts_recv[EAST]*(nparts_recv[EAST] > 0) + (nparts_recv[EAST] == 0);
  int recv_alloc_w = nparts_recv[WEST]*(nparts_recv[WEST] > 0) + (nparts_recv[WEST] == 0);

  checkCudaErrors(hipMalloc(&_send_parts_e, send_alloc_e * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_parts_w, send_alloc_w * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_e, recv_alloc_e * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_w, recv_alloc_w * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_s_parts_e, send_alloc_e * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_send_s_parts_w, send_alloc_w * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_e, recv_alloc_e * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_w, recv_alloc_w * sizeof(part_struct_scalar)));

  /* Pack particles into _send_parts */
  if (nparts_send[EAST] > 0) {
    pack_parts_e<<<bin_num_inb, bin_dim_inb>>>(_send_parts_e, _parts, _offset_e,
      _bin_start, _bin_count, _part_ind);
  } else {  // fill dummy data
    //hipMemset(_send_parts_e, 0., send_alloc_e * sizeof(part_struct));
  }

  if (nparts_send[WEST] > 0) {
    pack_parts_w<<<bin_num_inb, bin_dim_inb>>>(_send_parts_w, _parts, _offset_w,
      _bin_start, _bin_count, _part_ind);
  } else {  // fill dummy data
    //hipMemset(_send_parts_w, 0., send_alloc_w * sizeof(part_struct));
  }
  hipDeviceSynchronize(); // To ensure packing is complete before sending

  /* Communicate particles with MPI */
  mpi_send_parts_i();

  // send particle scalar information
  if (nparts_send[EAST] > 0) {
    pack_s_parts_e<<<bin_num_inb, bin_dim_inb>>>(_send_s_parts_e, _s_parts, _offset_e,
      _bin_start, _bin_count, _part_ind);
  }
  if (nparts_send[WEST] > 0) {
    pack_s_parts_w<<<bin_num_inb, bin_dim_inb>>>(_send_s_parts_w, _s_parts, _offset_w,
      _bin_start, _bin_count, _part_ind);
  }
  hipDeviceSynchronize();
  mpi_send_s_parts_i();

  /* Find number of particles currently in the EAST/WEST ghost bins */
  int nparts_ghost[6];

  if (nparts > 0) {
    // East
    offset = GFX_LOC(bins.Gcc._ieb, 0, 0, s1b, s2b);
    if (dom[rank].e != MPI_PROC_NULL) {
      nparts_ghost[EAST] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + bins.Gcc.s2b_i,
                                          0., thrust::plus<int>());
    } else {
      nparts_ghost[EAST] = 0;
    }

    // West
    offset = GFX_LOC(bins.Gcc._isb, 0, 0, s1b, s2b);
    if (dom[rank].w != MPI_PROC_NULL) {
      nparts_ghost[WEST] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + bins.Gcc.s2b_i,
                                          0., thrust::plus<int>());
    } else {
      nparts_ghost[WEST] = 0;
    }
  } else { // no parts
    nparts_ghost[EAST] = 0;
    nparts_ghost[WEST] = 0;
  }

  /* Calculate new number of particles */
  int nparts_old = nparts;
  nparts += nparts_recv[EAST] + nparts_recv[WEST] 
          - nparts_ghost[EAST] - nparts_ghost[WEST];

  /* allocate temporary part struct */
  part_struct *_tmp_parts;
  checkCudaErrors(hipMalloc(&_tmp_parts, nparts * sizeof(part_struct)));
  part_struct_scalar *_tmp_s_parts;
  checkCudaErrors(hipMalloc(&_tmp_s_parts, nparts * sizeof(part_struct_scalar)));

  if (nparts_old > 0) {
    /* parallel prefix scan of [_is, _ie] of _bin_count */
    int *_offset_all;
    checkCudaErrors(hipMalloc(&_offset_all, bins.Gcc.s3b * sizeof(int)));
    thrust::device_ptr<int> t_offset_all(_offset_all);

    // Scan over bin_count[_is->_ie, j, k]
    int size = bins.Gcc.s3b - 2*bins.Gcc.s2b_i;
    
    thrust::exclusive_scan(t_bin_count + bins.Gcc.s2b_i,
                           t_bin_count + bins.Gcc.s2b_i + size,
                           t_offset_all + bins.Gcc.s2b_i);

    /* copy bins of particles to tmp_parts */
    copy_central_bin_parts_i<<<bin_num_inb, bin_dim_inb>>>(_tmp_parts, _parts,
      _bin_start, _bin_count, _part_ind, _offset_all);
	copy_central_bin_s_parts_i<<<bin_num_inb, bin_dim_inb>>>(_tmp_s_parts, _s_parts,
      _bin_start, _bin_count, _part_ind, _offset_all);

    hipFree(_offset_all);

  } else { // no (old) parts
    // Do not need to copy or prefix scan
  }

  /* Copy ghost particles received from WEST */
  if (nparts_recv[WEST] > 0) {
    t_nparts = nparts_recv[WEST] * (nparts_recv[WEST] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[WEST] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[WEST] / (real) t_nparts);

    dim3 dim_nparts_w(t_nparts);
    dim3 num_nparts_w(b_nparts);

    offset = nparts_old - nparts_ghost[WEST] - nparts_ghost[EAST];
    copy_ghost_bin_parts<<<num_nparts_w, dim_nparts_w>>>(_tmp_parts, _recv_parts_w,
      nparts_recv[WEST], offset, WEST, _DOM);
	copy_ghost_bin_s_parts<<<num_nparts_w, dim_nparts_w>>>(_tmp_s_parts, _recv_s_parts_w,
      nparts_recv[WEST], offset, WEST, _DOM);
  } else { // nparts_recv[WEST] <= 0
    // Do nothing
  }

  /* Copy ghost particles received from EAST */
  if (nparts_recv[EAST] > 0) {
    t_nparts = nparts_recv[EAST] * (nparts_recv[EAST] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[EAST] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[EAST] / (real) t_nparts);

    dim3 dim_nparts_e(t_nparts);
    dim3 num_nparts_e(b_nparts);

    offset = nparts_old - nparts_ghost[WEST] - nparts_ghost[EAST] 
            + nparts_recv[WEST];
    copy_ghost_bin_parts<<<num_nparts_e, dim_nparts_e>>>(_tmp_parts, _recv_parts_e,
      nparts_recv[EAST], offset, EAST, _DOM);
    copy_ghost_bin_s_parts<<<num_nparts_e, dim_nparts_e>>>(_tmp_s_parts, _recv_s_parts_e,
      nparts_recv[EAST], offset, EAST, _DOM);
  } else { // npats_recv[EAST] <= 0
    // Do nothing
  }

  /* Swap pointers to _parts and _tmp_parts */
  part_struct *tmp = _parts;
  _parts = _tmp_parts;
  _tmp_parts = tmp;
  part_struct_scalar *s_tmp = _s_parts;
  _s_parts = _tmp_s_parts;
  _tmp_s_parts = s_tmp;

//  /* Correct ghost particle position for periodic boundaries */
//  int nparts_added = nparts_recv[EAST] + nparts_recv[WEST];
//  if (nparts_added > 0) {
//    t_nparts = nparts_added * (nparts_added < MAX_THREADS_1D)
//              + MAX_THREADS_1D * (nparts_added >= MAX_THREADS_1D);
//    b_nparts = (int) ceil((real) nparts_added / (real) t_nparts);
//
//    dim3 dim_nparts_a(t_nparts);
//    dim3 num_nparts_a(b_nparts);
//
//    offset = nparts_old - nparts_ghost[WEST] - nparts_ghost[EAST];
//    correct_periodic_boundaries_i<<<num_nparts_a, dim_nparts_a>>>(_parts, 
//      offset, nparts_added, _bc, _DOM);
//  }

  // Free memory
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_e);
  hipFree(_offset_w);
  hipFree(_send_parts_e);
  hipFree(_send_parts_w);
  hipFree(_recv_parts_e);
  hipFree(_recv_parts_w);
  hipFree(_send_s_parts_e);
  hipFree(_send_s_parts_w);
  hipFree(_recv_s_parts_e);
  hipFree(_recv_s_parts_w);
  hipFree(_tmp_parts);
  hipFree(_tmp_s_parts);
}

extern "C"
void cuda_scalar_transfer_parts_j(void)
{
  // Steps are the same as in cuda_transfer_part_i, except we index with 'j'
  // varying the slowest

  /* Init execution config */

  // thread over north/south faces 
  int tz = bins.Gcc.knb * (bins.Gcc.knb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.knb >= MAX_THREADS_DIM);
  int tx = bins.Gcc.inb * (bins.Gcc.inb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.inb >= MAX_THREADS_DIM);

  int bz = (int) ceil((real) bins.Gcc.knb / (real) tz);
  int bx = (int) ceil((real) bins.Gcc.inb / (real) tx);

  dim3 bin_num_jnb(bz, bx);
  dim3 bin_dim_jnb(tz, tx);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b, s2b; // custom strides
  int offset;

  /* Allocate memory */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_n;
  int *_offset_s;
  checkCudaErrors(hipMalloc(&_offset_n, bins.Gcc.s2b_j * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_s, bins.Gcc.s2b_j * sizeof(int)));
  thrust::device_ptr<int> t_offset_n(_offset_n);
  thrust::device_ptr<int> t_offset_s(_offset_s);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  // If we have parts...
  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_j<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_j<<<bin_num_jnb, bin_dim_jnb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send, and packing offsets */
    s1b = bins.Gcc.knb;
    s2b = s1b * bins.Gcc.inb;
  
    // North
    offset = GFY_LOC(0, bins.Gcc._je, 0, s1b, s2b);
    if (dom[rank].n != MPI_PROC_NULL) {
      // _bin_count is indexed with j varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _je plane
      nparts_send[NORTH] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + bins.Gcc.s2b_j,
                                          0., thrust::plus<int>());

      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[NORTH] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_j, t_offset_n);
      } else {
        hipMemset(_offset_n, 0., bins.Gcc.s2b_j * sizeof(int));
      }
      
    } else {
      nparts_send[NORTH] = 0;
      hipMemset(_offset_n, 0., bins.Gcc.s2b_j * sizeof(int));
    }
  
    // South
    offset = GFY_LOC(0, bins.Gcc._js, 0, s1b, s2b);
    if (dom[rank].s != MPI_PROC_NULL) {
      nparts_send[SOUTH] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + bins.Gcc.s2b_j,
                                          0., thrust::plus<int>());

      if (nparts_send[SOUTH] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_j, t_offset_s);
      } else {
        hipMemset(_offset_s, 0., bins.Gcc.s2b_j * sizeof(int));
      }

    } else {
      nparts_send[SOUTH] = 0;
      hipMemset(_offset_s, 0., bins.Gcc.s2b_j * sizeof(int));
    }
  
  } else { // nparts <= 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[NORTH] = 0;
    nparts_send[SOUTH] = 0;
    hipMemset(_offset_n, 0., bins.Gcc.s2b_j * sizeof(int));
    hipMemset(_offset_s, 0., bins.Gcc.s2b_j * sizeof(int));
  }

  /* Send number of parts to north/south */
  nparts_recv[SOUTH] = 0; // init
  nparts_recv[NORTH] = 0;
  mpi_send_nparts_j();

  /* Allocate memory for send and receiving particles */
  // See accompanying note at the same location in cuda_transfer_parts_i
  int send_alloc_n = nparts_send[NORTH]*(nparts_send[NORTH] > 0) + (nparts_send[NORTH] == 0);
  int send_alloc_s = nparts_send[SOUTH]*(nparts_send[SOUTH] > 0) + (nparts_send[SOUTH] == 0);
  int recv_alloc_n = nparts_recv[NORTH]*(nparts_recv[NORTH] > 0) + (nparts_recv[NORTH] == 0);
  int recv_alloc_s = nparts_recv[SOUTH]*(nparts_recv[SOUTH] > 0) + (nparts_recv[SOUTH] == 0);

  checkCudaErrors(hipMalloc(&_send_parts_n, send_alloc_n * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_parts_s, send_alloc_s * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_n, recv_alloc_n * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_s, recv_alloc_s * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_s_parts_n, send_alloc_n * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_send_s_parts_s, send_alloc_s * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_n, recv_alloc_n * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_s, recv_alloc_s * sizeof(part_struct_scalar)));

  /* Pack particles into _send_parts */
  if (nparts_send[NORTH] > 0)  {
    pack_parts_n<<<bin_num_jnb, bin_dim_jnb>>>(_send_parts_n, _parts, _offset_n,
      _bin_start, _bin_count, _part_ind);
  } else { // fill dummy data
    //hipMemset(_send_parts_n, 0., send_alloc_n * sizeof(part_struct));
  }

  if (nparts_send[SOUTH] > 0)  {
    pack_parts_s<<<bin_num_jnb, bin_dim_jnb>>>(_send_parts_s, _parts, _offset_s,
      _bin_start, _bin_count, _part_ind);
  } else { // fill dummy data
    //hipMemset(_send_parts_s, 0., send_alloc_s * sizeof(part_struct));
  }
  hipDeviceSynchronize(); // To ensure packing is complete before sending

  /* Communicate particles with MPI */
  mpi_send_parts_j();

  // send particle scalar information
  if (nparts_send[NORTH] > 0) {
    pack_s_parts_n<<<bin_num_jnb, bin_dim_jnb>>>(_send_s_parts_n, _s_parts, _offset_n,
      _bin_start, _bin_count, _part_ind);
  }
  if (nparts_send[SOUTH] > 0) {
    pack_s_parts_s<<<bin_num_jnb, bin_dim_jnb>>>(_send_s_parts_s, _s_parts, _offset_s,
      _bin_start, _bin_count, _part_ind);
  }
  hipDeviceSynchronize();
  mpi_send_s_parts_j();

  /* Find number of particles currently in the NORTH/SOUTH ghost bins */
  int nparts_ghost[6];

  if (nparts > 0) {
    // North
    offset = GFY_LOC(0, bins.Gcc._jeb, 0, s1b, s2b);
    if (dom[rank].n != MPI_PROC_NULL) {
      nparts_ghost[NORTH] = thrust::reduce(t_bin_count + offset,
                                           t_bin_count + offset + bins.Gcc.s2b_j,
                                           0., thrust::plus<int>());
    } else {
      nparts_ghost[NORTH] = 0;
    }

    // South
    offset = GFY_LOC(0, bins.Gcc._jsb, 0, s1b, s2b);
    if (dom[rank].s != MPI_PROC_NULL) {
      nparts_ghost[SOUTH] = thrust::reduce(t_bin_count + offset,
                                           t_bin_count + offset + bins.Gcc.s2b_j,
                                           0., thrust::plus<int>());
    } else {
      nparts_ghost[SOUTH] = 0;
    }
  } else { // no parts
    nparts_ghost[NORTH] = 0;
    nparts_ghost[SOUTH] = 0;
  }

  /* Calculate new number of particles */
  int nparts_old = nparts;
  nparts += nparts_recv[NORTH] + nparts_recv[SOUTH] 
          - nparts_ghost[NORTH] - nparts_ghost[SOUTH];

  /* allocate temporary part struct */
  part_struct *_tmp_parts;
  checkCudaErrors(hipMalloc(&_tmp_parts, nparts * sizeof(part_struct)));
  part_struct_scalar *_tmp_s_parts;
  checkCudaErrors(hipMalloc(&_tmp_s_parts, nparts * sizeof(part_struct_scalar)));

  if (nparts_old > 0) {
    /* parallel prefix scan of ALL of _bin_count */
    int *_offset_all;
    checkCudaErrors(hipMalloc(&_offset_all, bins.Gcc.s3b * sizeof(int)));

    // Scan over bin_count[i, _js->_je, k]
    int size = bins.Gcc.s3b - 2*bins.Gcc.s2b_j;
    thrust::device_ptr<int> t_offset_all(_offset_all);
    thrust::exclusive_scan(t_bin_count + bins.Gcc.s2b_j,
                           t_bin_count + bins.Gcc.s2b_j + size,
                           t_offset_all + bins.Gcc.s2b_j);


    /* copy bins of particles to tmp_parts */
    copy_central_bin_parts_j<<<bin_num_jnb, bin_dim_jnb>>>(_tmp_parts, _parts,
      _bin_start, _bin_count, _part_ind, _offset_all);
    copy_central_bin_s_parts_j<<<bin_num_jnb, bin_dim_jnb>>>(_tmp_s_parts, _s_parts,
      _bin_start, _bin_count, _part_ind, _offset_all);

    hipFree(_offset_all);

  } else { // no (old) parts
    // Do nothing
  }

  /* Copy ghost particles recieved from SOUTH */
  if (nparts_recv[SOUTH] > 0) {
    t_nparts = nparts_recv[SOUTH] * (nparts_recv[SOUTH] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[SOUTH] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[SOUTH] / (real) t_nparts);

    dim3 dim_nparts_s(t_nparts);
    dim3 num_nparts_s(b_nparts);

    offset = nparts_old - nparts_ghost[SOUTH] - nparts_ghost[NORTH];
    copy_ghost_bin_parts<<<num_nparts_s, dim_nparts_s>>>(_tmp_parts, _recv_parts_s,
      nparts_recv[SOUTH], offset, SOUTH, _DOM);
    copy_ghost_bin_s_parts<<<num_nparts_s, dim_nparts_s>>>(_tmp_s_parts, _recv_s_parts_s,
      nparts_recv[SOUTH], offset, SOUTH, _DOM);
  } else { // nparts_recv[SOUTH] <= 0
    // Do nothing
  }

  /* Copy ghost particles received from NORTH */
  if (nparts_recv[NORTH] > 0) {
    t_nparts = nparts_recv[NORTH] * (nparts_recv[NORTH] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[NORTH] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[NORTH] / (real) t_nparts);

    dim3 dim_nparts_n(t_nparts);
    dim3 num_nparts_n(b_nparts);

    offset = nparts_old - nparts_ghost[SOUTH] - nparts_ghost[NORTH]
            + nparts_recv[SOUTH];
    copy_ghost_bin_parts<<<num_nparts_n, dim_nparts_n>>>(_tmp_parts, _recv_parts_n,
      nparts_recv[NORTH], offset, NORTH, _DOM);
    copy_ghost_bin_s_parts<<<num_nparts_n, dim_nparts_n>>>(_tmp_s_parts, _recv_s_parts_n,
      nparts_recv[NORTH], offset, NORTH, _DOM);
  } else { // nparts_recv[NORTH] <= 0
    // Do nothing
  }

  /* Swap pointers to _parts and _tmp_parts */
  part_struct *tmp = _parts;
  _parts = _tmp_parts;
  _tmp_parts = tmp;
  part_struct_scalar *s_tmp = _s_parts;
  _s_parts = _tmp_s_parts;
  _tmp_s_parts = s_tmp;

//  /* Correct ghost particle position for periodic boundaries */
//  int nparts_added = nparts_recv[NORTH] + nparts_recv[SOUTH];
//  if (nparts_added > 0) {
//    t_nparts = nparts_added * (nparts_added < MAX_THREADS_1D)
//              + MAX_THREADS_1D * (nparts_added >= MAX_THREADS_1D);
//    b_nparts = (int) ceil((real) nparts_added / (real) t_nparts);
//
//    dim3 dim_nparts_a(t_nparts);
//    dim3 num_nparts_a(b_nparts);
//
//    offset = nparts_old - nparts_ghost[SOUTH] - nparts_ghost[NORTH];
//    correct_periodic_boundaries_j<<<num_nparts_a, dim_nparts_a>>>(_parts, 
//      offset, nparts_added, _bc, _DOM);
//  }

  // Free memory
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_n);
  hipFree(_offset_s);
  hipFree(_send_parts_n);
  hipFree(_send_parts_s);
  hipFree(_recv_parts_n);
  hipFree(_recv_parts_s);
  hipFree(_send_s_parts_n);
  hipFree(_send_s_parts_s);
  hipFree(_recv_s_parts_n);
  hipFree(_recv_s_parts_s);
  hipFree(_tmp_parts);
  hipFree(_tmp_s_parts);
}

extern "C"
void cuda_scalar_transfer_parts_k(void)
{
  // Steps are the same as in cuda_transfer_part_i, except we index with 'k'
  // varying the slowest

  /* Init execution config */

  // thread over top/bottom faces 
  int tx = bins.Gcc.inb * (bins.Gcc.inb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.inb >= MAX_THREADS_DIM);
  int ty = bins.Gcc.jnb * (bins.Gcc.jnb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.jnb >= MAX_THREADS_DIM);

  int bx = (int) ceil((real) bins.Gcc.inb / (real) tx);
  int by = (int) ceil((real) bins.Gcc.jnb / (real) ty);

  dim3 bin_num_knb(bx, by);
  dim3 bin_dim_knb(tx, ty);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b = bins.Gcc.inb;
  int s2b = s1b * bins.Gcc.jnb;
  int offset;

  /* Allocate memory */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_t;
  int *_offset_b;
  checkCudaErrors(hipMalloc(&_offset_t, bins.Gcc.s2b_k * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_b, bins.Gcc.s2b_k * sizeof(int)));
  thrust::device_ptr<int> t_offset_t(_offset_t);
  thrust::device_ptr<int> t_offset_b(_offset_b);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  // If we have parts...
  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_k<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }
    //_part_bin = thrust::raw_pointer_cast(t_part_bin);
    //_part_ind = thrust::raw_pointer_cast(t_part_ind);

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_k<<<bin_num_knb, bin_dim_knb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send, and packing offsets */
    // Top
    offset = GFZ_LOC(0, 0, bins.Gcc._ke, s1b, s2b);
    if (dom[rank].t != MPI_PROC_NULL) {
      // _bin_count is indexed with k varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _ke plane
      nparts_send[TOP] = thrust::reduce(t_bin_count + offset,
                                        t_bin_count + offset + bins.Gcc.s2b_k,
                                          0., thrust::plus<int>());
    
      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[TOP] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_k, t_offset_t);
      } else {
        hipMemset(_offset_t, 0., bins.Gcc.s2b_k * sizeof(int));
      }

    } else {
      nparts_send[TOP] = 0;
      hipMemset(_offset_t, 0., bins.Gcc.s2b_k * sizeof(int));
    }

    // Bottom
    offset = GFZ_LOC(0, 0, bins.Gcc._ks, s1b, s2b);
    if (dom[rank].b != MPI_PROC_NULL) {
      nparts_send[BOTTOM] = thrust::reduce(t_bin_count + offset,
                                           t_bin_count + offset + bins.Gcc.s2b_k,
                                           0., thrust::plus<int>());

      if (nparts_send[BOTTOM] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + bins.Gcc.s2b_k, t_offset_b);
      } else {
        hipMemset(_offset_b, 0., bins.Gcc.s2b_k * sizeof(int));
      }

    } else {
      nparts_send[BOTTOM] = 0;
      hipMemset(_offset_b, 0., bins.Gcc.s2b_k * sizeof(int));
    }
    
  } else { // nparts <= 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[TOP] = 0;
    nparts_send[BOTTOM] = 0;
    hipMemset(_offset_t, 0., bins.Gcc.s2b_k * sizeof(int));
    hipMemset(_offset_b, 0., bins.Gcc.s2b_k * sizeof(int));
  }

  /* Send number of parts to top/bottom */
  nparts_recv[TOP] = 0; // init
  nparts_recv[BOTTOM] = 0;
  mpi_send_nparts_k();

  /* Allocate memory for send and receiving particles */
  // See accompanying note at the same location in cuda_transfer_parts_i
  int send_alloc_t = nparts_send[TOP]*(nparts_send[TOP] > 0) + (nparts_send[TOP] == 0);
  int send_alloc_b = nparts_send[BOTTOM]*(nparts_send[BOTTOM] > 0) + (nparts_send[BOTTOM] == 0);
  int recv_alloc_t = nparts_recv[TOP]*(nparts_recv[TOP] > 0) + (nparts_recv[TOP] == 0);
  int recv_alloc_b = nparts_recv[BOTTOM]*(nparts_recv[BOTTOM] > 0) + (nparts_recv[BOTTOM] == 0);

  checkCudaErrors(hipMalloc(&_send_parts_t, send_alloc_t * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_parts_b, send_alloc_b * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_t, recv_alloc_t * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_recv_parts_b, recv_alloc_b * sizeof(part_struct)));
  checkCudaErrors(hipMalloc(&_send_s_parts_t, send_alloc_t * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_send_s_parts_b, send_alloc_b * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_t, recv_alloc_t * sizeof(part_struct_scalar)));
  checkCudaErrors(hipMalloc(&_recv_s_parts_b, recv_alloc_b * sizeof(part_struct_scalar)));

  /* Pack particles into _send_parts */
  if (nparts_send[TOP] > 0) {
    pack_parts_t<<<bin_num_knb, bin_dim_knb>>>(_send_parts_t, _parts, _offset_t,
      _bin_start, _bin_count, _part_ind);
  } else {  // fill dummy data
    //hipMemset(_send_parts_t, 0., send_alloc_t * sizeof(part_struct));
  }

  if (nparts_send[BOTTOM] > 0) {
    pack_parts_b<<<bin_num_knb, bin_dim_knb>>>(_send_parts_b, _parts, _offset_b,
      _bin_start, _bin_count, _part_ind);
  } else {  // fill dummy data
    //hipMemset(_send_parts_b, 0., send_alloc_b * sizeof(part_struct));
  }
  hipDeviceSynchronize(); // To ensure packing is complete before sending

  /* Communicate particles with MPI */
  mpi_send_parts_k();

  // send particle scalar information
  if (nparts_send[TOP] > 0) {
    pack_s_parts_t<<<bin_num_knb, bin_dim_knb>>>(_send_s_parts_t, _s_parts, _offset_t,
      _bin_start, _bin_count, _part_ind);
  }
  if (nparts_send[BOTTOM] > 0) {
    pack_s_parts_b<<<bin_num_knb, bin_dim_knb>>>(_send_s_parts_b, _s_parts, _offset_b,
      _bin_start, _bin_count, _part_ind);
  }
  hipDeviceSynchronize();
  mpi_send_s_parts_k();

  /* Find number of particles currently in the TOP/BOTTOM ghost bins */
  int nparts_ghost[6];
  
  if (nparts > 0) {
    // TOP
    offset = GFZ_LOC(0, 0, bins.Gcc._keb, s1b, s2b);
    if (dom[rank].t != MPI_PROC_NULL) {
      nparts_ghost[TOP] = thrust::reduce(t_bin_count + offset,
                                         t_bin_count + offset + bins.Gcc.s2b_k,
                                         0., thrust::plus<int>());
    } else {
      nparts_ghost[TOP] = 0;
    }

    // BOTTOM
    offset = GFZ_LOC(0, 0, bins.Gcc._ksb, s1b, s2b);
    if (dom[rank].b != MPI_PROC_NULL) {
      nparts_ghost[BOTTOM] = thrust::reduce(t_bin_count + offset,
                                            t_bin_count + offset + bins.Gcc.s2b_k,
                                            0., thrust::plus<int>());
    } else {
      nparts_ghost[BOTTOM] = 0;
    }
  } else { // no parts
    nparts_ghost[TOP] = 0;
    nparts_ghost[BOTTOM] = 0;
  }

  /* Calculate new number of particles */
  int nparts_old = nparts;
  nparts += nparts_recv[TOP] + nparts_recv[BOTTOM] 
          - nparts_ghost[TOP] - nparts_ghost[BOTTOM];

  /* allocate temporary part struct */
  part_struct *_tmp_parts;
  checkCudaErrors(hipMalloc(&_tmp_parts, nparts * sizeof(part_struct)));
  part_struct_scalar *_tmp_s_parts;
  checkCudaErrors(hipMalloc(&_tmp_s_parts, nparts * sizeof(part_struct_scalar)));

  if (nparts_old > 0) {
    /* parallel prefix scan of ALL of _bin_count */
    int *_offset_all;
    checkCudaErrors(hipMalloc(&_offset_all, bins.Gcc.s3b * sizeof(int)));

    // Scan over bin_count[i, m, _ks->_ke]
    int size = bins.Gcc.s3b - 2*bins.Gcc.s2b_k;
    thrust::device_ptr<int> t_offset_all(_offset_all);
    thrust::exclusive_scan(t_bin_count + bins.Gcc.s2b_k,
                           t_bin_count + bins.Gcc.s2b_k + size,
                           t_offset_all + bins.Gcc.s2b_k);


    /* copy bins of particles to tmp_parts */
    copy_central_bin_parts_k<<<bin_num_knb, bin_dim_knb>>>(_tmp_parts, _parts,
      _bin_start, _bin_count, _part_ind, _offset_all);
    copy_central_bin_s_parts_k<<<bin_num_knb, bin_dim_knb>>>(_tmp_s_parts, _s_parts,
      _bin_start, _bin_count, _part_ind, _offset_all);

    hipFree(_offset_all);

  } else { // no (old) parts
    // Do nothing
  }

  /* Copy ghost particles recieved from BOTTOM */
  if (nparts_recv[BOTTOM] > 0) {
    t_nparts = nparts_recv[BOTTOM] * (nparts_recv[BOTTOM] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[BOTTOM] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[BOTTOM] / (real) t_nparts);

    dim3 dim_nparts_b(t_nparts);
    dim3 num_nparts_b(b_nparts);

    offset = nparts_old - nparts_ghost[BOTTOM] - nparts_ghost[TOP];
    copy_ghost_bin_parts<<<num_nparts_b, dim_nparts_b>>>(_tmp_parts, _recv_parts_b,
      nparts_recv[BOTTOM], offset, BOTTOM, _DOM);
    copy_ghost_bin_s_parts<<<num_nparts_b, dim_nparts_b>>>(_tmp_s_parts, _recv_s_parts_b,
      nparts_recv[BOTTOM], offset, BOTTOM, _DOM);
  } else { // nparts_recv[BOTTOM] <= 0
    // Do nothing
  }

  /* Copy ghost particles received from TOP */
  if (nparts_recv[TOP] > 0) {
    t_nparts = nparts_recv[TOP] * (nparts_recv[TOP] < MAX_THREADS_1D)
              + MAX_THREADS_1D * (nparts_recv[TOP] >= MAX_THREADS_1D);
    b_nparts = (int) ceil((real) nparts_recv[TOP] / (real) t_nparts);

    dim3 dim_nparts_t(t_nparts);
    dim3 num_nparts_t(b_nparts);

    offset = nparts_old - nparts_ghost[BOTTOM] - nparts_ghost[TOP]
            + nparts_recv[BOTTOM];
    copy_ghost_bin_parts<<<num_nparts_t, dim_nparts_t>>>(_tmp_parts, _recv_parts_t,
      nparts_recv[TOP], offset, TOP, _DOM);
    copy_ghost_bin_s_parts<<<num_nparts_t, dim_nparts_t>>>(_tmp_s_parts, _recv_s_parts_t,
      nparts_recv[TOP], offset, TOP, _DOM);
  } else { // nparts_recv[TOP] <= 0
    // Do nothing
  }

  /* Swap pointers to _parts and _tmp_parts */
  part_struct *tmp = _parts;
  _parts = _tmp_parts;
  _tmp_parts = tmp;
  part_struct_scalar *s_tmp = _s_parts;
  _s_parts = _tmp_s_parts;
  _tmp_s_parts = s_tmp;

//  /* Correct ghost particle position for periodic boundaries */
//  int nparts_added = nparts_recv[TOP] + nparts_recv[BOTTOM];
//  if (nparts_added > 0) {
//    t_nparts = nparts_added * (nparts_added < MAX_THREADS_1D)
//              + MAX_THREADS_1D * (nparts_added >= MAX_THREADS_1D);
//    b_nparts = (int) ceil((real) nparts_added / (real) t_nparts);
//
//    dim3 dim_nparts_a(t_nparts);
//    dim3 num_nparts_a(b_nparts);
//
//    offset = nparts_old - nparts_ghost[BOTTOM] - nparts_ghost[TOP];
//    correct_periodic_boundaries_k<<<num_nparts_a, dim_nparts_a>>>(_parts, 
//      offset, nparts_added, _bc, _DOM);
//   
//  }

  // Free memory
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_t);
  hipFree(_offset_b);
  hipFree(_send_parts_t);
  hipFree(_send_parts_b);
  hipFree(_recv_parts_t);
  hipFree(_recv_parts_b);
  hipFree(_send_s_parts_t);
  hipFree(_send_s_parts_b);
  hipFree(_recv_s_parts_t);
  hipFree(_recv_s_parts_b);
  hipFree(_tmp_parts);
  hipFree(_tmp_s_parts);
}

extern "C"
void cuda_scalar_malloc_host(void)
{
  checkCudaErrors(hipHostMalloc(&s, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipHostMalloc(&s0, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipHostMalloc(&s_conv, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipHostMalloc(&s_conv0, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipHostMalloc(&s_diff, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipHostMalloc(&s_diff0, dom[rank].Gcc.s3b * sizeof(real)));
    cpumem += dom[rank].Gcc.s3b * sizeof(real);
}

extern "C"
void cuda_scalar_malloc_dev(void)
{
  checkCudaErrors(hipMalloc(&_bc_s, sizeof(BC_s)));
    gpumem += sizeof(BC_s);
  checkCudaErrors(hipMemcpy(_bc_s, &bc_s, sizeof(BC_s), 
    hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&_s, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipMalloc(&_s0, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipMalloc(&_s_conv, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipMalloc(&_s_conv0, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipMalloc(&_s_diff, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);
  checkCudaErrors(hipMalloc(&_s_diff0, dom[rank].Gcc.s3b * sizeof(real)));
    gpumem += dom[rank].Gcc.s3b * sizeof(real);

  checkCudaErrors(hipMemset(_s, 0., dom[rank].Gcc.s3b * sizeof(real)));
  checkCudaErrors(hipMemset(_s0, 0., dom[rank].Gcc.s3b * sizeof(real)));
  checkCudaErrors(hipMemset(_s_conv, 0., dom[rank].Gcc.s3b * sizeof(real)));
  checkCudaErrors(hipMemset(_s_conv0, 0., dom[rank].Gcc.s3b * sizeof(real)));
  checkCudaErrors(hipMemset(_s_diff, 0., dom[rank].Gcc.s3b * sizeof(real)));
  checkCudaErrors(hipMemset(_s_diff0, 0., dom[rank].Gcc.s3b * sizeof(real)));
}

extern "C"
void cuda_scalar_push(void)
{
  checkCudaErrors(hipMemcpy(_s, s, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(_s0, s0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(_s_conv, s_conv, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(_s_conv0, s_conv0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(_s_diff, s_diff, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(_s_diff0, s_diff0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyHostToDevice));
}

extern "C"
void cuda_scalar_pull(void)
{
  checkCudaErrors(hipMemcpy(s, _s, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
}

extern "C"
void cuda_scalar_pull_debug(void)
{
  checkCudaErrors(hipMemcpy(s_conv, _s_conv, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(s_diff, _s_diff, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
}

extern "C"
void cuda_scalar_pull_restart(void)
{
  checkCudaErrors(hipMemcpy(s0, _s0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(s_conv0, _s_conv0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(s_diff0, _s_diff0, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToHost));
}

extern "C"
void cuda_scalar_free(void)
{
    checkCudaErrors(hipHostFree(s));
    checkCudaErrors(hipHostFree(s0));
    checkCudaErrors(hipHostFree(s_conv));
    checkCudaErrors(hipHostFree(s_conv0));
    checkCudaErrors(hipHostFree(s_diff));
    checkCudaErrors(hipHostFree(s_diff0));

    checkCudaErrors(hipFree(_bc_s));

    checkCudaErrors(hipFree(_s));
    checkCudaErrors(hipFree(_s0));
    checkCudaErrors(hipFree(_s_conv));
    checkCudaErrors(hipFree(_s_conv0));
    checkCudaErrors(hipFree(_s_diff));
    checkCudaErrors(hipFree(_s_diff0));
}

extern "C"
void cuda_scalar_part_malloc_dev(void)
{
  if (NPARTS > 0 && SCALAR >= 1) {
    checkCudaErrors(hipMalloc(&_s_parts, nparts * sizeof(part_struct_scalar)));
    gpumem += nparts * sizeof(part_struct_scalar);
  }
}

extern "C"
void cuda_scalar_part_push(void)
{
  if (NPARTS > 0 && SCALAR >= 1) {
    checkCudaErrors(hipMemcpy(_s_parts, s_parts, nparts * sizeof(part_struct_scalar),
      hipMemcpyHostToDevice));

    int s_nn[64] = {0,
                  1, 1, 1,
                  2, 2, 2, 2, 2,
                  3, 3, 3, 3, 3, 3, 3,
                  4, 4, 4, 4, 4, 4, 4, 4, 4,
                  5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5,
                  6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6,
                  7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7};

    int s_mm[64] = {0,
                  -1, 0, 1,
                  -2, -1, 0, 1, 2,
                  -3, -2, -1, 0, 1, 2, 3,
                  -4, -3, -2, -1, 0, 1, 2, 3, 4,
                  -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5,
                  -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6,
                  -7, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, 7};

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_s_mm), s_mm, 64 * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_s_nn), s_nn, 64 * sizeof(int)));
  }
}

extern "C"
void cuda_scalar_part_pull_with_scalar(void)
{
  /* Declare temporary part structure and nparts_subdom */
  part_struct *_tmp_parts;
  part_struct_scalar *_tmp_s_parts;
  nparts_subdom = 0;

  /* Re-allocate memory */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  if (nparts > 0) {
    // Thread over nparts
    int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                  + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
    int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

    dim3 dim_nparts(t_nparts);
    dim3 num_nparts(b_nparts);

    // thread over top/bottom faces 
    int tx = bins.Gcc.inb * (bins.Gcc.inb < MAX_THREADS_DIM)
         + MAX_THREADS_DIM * (bins.Gcc.inb >= MAX_THREADS_DIM);
    int ty = bins.Gcc.jnb * (bins.Gcc.jnb < MAX_THREADS_DIM)
         + MAX_THREADS_DIM * (bins.Gcc.jnb >= MAX_THREADS_DIM);
    int tz = bins.Gcc.knb * (bins.Gcc.knb < MAX_THREADS_DIM)
         + MAX_THREADS_DIM * (bins.Gcc.knb >= MAX_THREADS_DIM);

    int bx = (int) ceil((real) bins.Gcc.inb / (real) tx);
    int by = (int) ceil((real) bins.Gcc.jnb / (real) ty);
    int bz = (int) ceil((real) bins.Gcc.knb / (real) tz);

    dim3 bin_num_inb(by, bz);
    dim3 bin_dim_inb(ty, tz);
    dim3 bin_num_jnb(bz, bx);
    dim3 bin_dim_jnb(tz, tx);
    dim3 bin_num_knb(bx, by);
    dim3 bin_dim_knb(tx, ty);

    /* Find each particle's bin */
    bin_fill_k<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_k<<<bin_num_knb, bin_dim_knb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Set ghost bin count to zero (GFZ indexed) */
    zero_ghost_bins_i<<<bin_num_inb, bin_dim_inb>>>(_bin_count);
    zero_ghost_bins_j<<<bin_num_jnb, bin_dim_jnb>>>(_bin_count);
    zero_ghost_bins_k<<<bin_num_knb, bin_dim_knb>>>(_bin_count);

    /* Allocate memory to find bin offset target indices in tmp part_struct */
    int *_bin_offset;
    checkCudaErrors(hipMalloc(&_bin_offset, bins.Gcc.s3b * sizeof(int)));

    /* Prefix scan _bin_count to find target indices in tmp part_struct */
    thrust::device_ptr<int> t_bin_count(_bin_count);
    thrust::device_ptr<int> t_bin_offset(_bin_offset);
    thrust::exclusive_scan(t_bin_count, t_bin_count + bins.Gcc.s3b, t_bin_offset);

    /* Reduce bin_count to find nparts in subdomain (ghost bins are zero'd) */
    nparts_subdom = thrust::reduce(t_bin_count, t_bin_count + bins.Gcc.s3b,
                                        0., thrust::plus<int>());

    /* Allocate new device part struct (no ghost particles) */
    checkCudaErrors(hipMalloc(&_tmp_parts, nparts_subdom * sizeof(part_struct)));
    checkCudaErrors(hipMalloc(&_tmp_s_parts, nparts_subdom * sizeof(part_struct_scalar)));

    /* Copy subdom parts to tmp part_struct (only in subdom, so [in, jn]) */
    // thread over inner bins (no ghost bins)
    tx = bins.Gcc.in * (bins.Gcc.in < MAX_THREADS_DIM)
     + MAX_THREADS_DIM * (bins.Gcc.in >= MAX_THREADS_DIM);
    ty = bins.Gcc.jn * (bins.Gcc.jn < MAX_THREADS_DIM)
     + MAX_THREADS_DIM * (bins.Gcc.jn >= MAX_THREADS_DIM);
    bx = (int) ceil((real) bins.Gcc.in / (real) tx);
    by = (int) ceil((real) bins.Gcc.jn / (real) ty);
    dim3 bin_num_kn(bx, by);
    dim3 bin_dim_kn(tx, ty);

    copy_subdom_parts_with_scalar<<<bin_num_kn, bin_dim_kn>>>(_tmp_parts, _parts,
      _tmp_s_parts, _s_parts, _bin_start, _bin_count, _part_ind, _bin_offset);

    hipFree(_bin_offset);

  } else { // nparts <= 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_subdom = 0;
    checkCudaErrors(hipMalloc(&_tmp_parts, nparts_subdom * sizeof(part_struct)));
    checkCudaErrors(hipMalloc(&_tmp_s_parts, nparts_subdom * sizeof(part_struct_scalar)));
  }

  /* Allocate new host parts with nparts in subdom */
  free(parts);
  free(s_parts);
  parts = (part_struct*) malloc(nparts_subdom * sizeof(part_struct));
  s_parts = (part_struct_scalar*) malloc(nparts_subdom * sizeof(part_struct_scalar));

  // Pull from device
  checkCudaErrors(hipMemcpy(parts, _tmp_parts, nparts_subdom * sizeof(part_struct),
    hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(s_parts, _tmp_s_parts, nparts_subdom * sizeof(part_struct_scalar),
    hipMemcpyDeviceToHost));

  // Free
  hipFree(_tmp_parts);
  hipFree(_tmp_s_parts);
  hipFree(_part_ind);
  hipFree(_part_bin);

  // Double check the number of particles is correct
  int reduce_parts = 0;
  MPI_Allreduce(&nparts_subdom, &reduce_parts, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
  if (reduce_parts != NPARTS) {
    printf("N%d >> Something went wrong. NPARTS = %d, but %d exist\n",
      rank, NPARTS, reduce_parts);
    printf("N%d >> Has %d parts\n", rank, nparts_subdom);
    exit(EXIT_FAILURE);
  }
}

extern "C"
void cuda_scalar_part_free(void)
{
  if(SCALAR >= 1 && NPARTS > 0) {
    checkCudaErrors(hipFree(_s_parts));
  }
}

extern "C"
void cuda_compute_boussinesq(void)
{
  forcing_boussinesq_x<<<blocks.Gfx.num_inb, blocks.Gfx.dim_inb>>>(s_alpha, g.x, s_init, _s, _f_x);
  forcing_boussinesq_y<<<blocks.Gfy.num_jnb, blocks.Gfy.dim_jnb>>>(s_alpha, g.y, s_init, _s, _f_y);
  forcing_boussinesq_z<<<blocks.Gfz.num_knb, blocks.Gfz.dim_knb>>>(s_alpha, g.z, s_init, _s, _f_z);
}

extern "C"
void cuda_scalar_BC(real *array)
{
  // Check whether each subdom boundary is an external boundary, then
  // apply the correct boundary conditions to all fields on that face

  // Only apply boundary conditions on the inner [*n x *n] plane, not the
  //  [*nb x *nb] -- this ensures we don't set the points that don't contain
  //  any solution, and we also don't set points twice

  /* WEST */
  if (dom[rank].w == MPI_PROC_NULL) {
    switch (bc_s.sW) {
      case DIRICHLET:
        BC_s_W_D<<<blocks.Gcc.num_in, blocks.Gcc.dim_in>>>(array, bc_s.sWD);
        break;
      case NEUMANN:
        BC_s_W_N<<<blocks.Gcc.num_in, blocks.Gcc.dim_in>>>(array, bc_s.sWN);
        break;
    }
  }

  /* EAST */
  if (dom[rank].e == MPI_PROC_NULL) {
    switch (bc_s.sE) {
      case DIRICHLET:
        BC_s_E_D<<<blocks.Gcc.num_in, blocks.Gcc.dim_in>>>(array, bc_s.sED);
        break;
      case NEUMANN:
        BC_s_E_N<<<blocks.Gcc.num_in, blocks.Gcc.dim_in>>>(array, bc_s.sEN);
        break;
    }
  }

  /* SOUTH */
  if (dom[rank].s == MPI_PROC_NULL) {
    switch (bc_s.sS) {
      case DIRICHLET:
        BC_s_S_D<<<blocks.Gcc.num_jn, blocks.Gcc.dim_jn>>>(array, bc_s.sSD);
        break;
      case NEUMANN:
        BC_s_S_N<<<blocks.Gcc.num_jn, blocks.Gcc.dim_jn>>>(array, bc_s.sSN);
        break;
    }
  }

  /* NORTH */
  if (dom[rank].n == MPI_PROC_NULL) {
    switch (bc_s.sN) {
      case DIRICHLET:
        BC_s_N_D<<<blocks.Gcc.num_jn, blocks.Gcc.dim_jn>>>(array, bc_s.sND);
        break;
      case NEUMANN:
        BC_s_N_N<<<blocks.Gcc.num_jn, blocks.Gcc.dim_jn>>>(array, bc_s.sNN);
        break;
    }
  }

  /* BOTTOM */
  if (dom[rank].b == MPI_PROC_NULL) {
    switch (bc_s.sB) {
      case DIRICHLET:
        BC_s_B_D<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(array, bc_s.sBD);
        break;
      case NEUMANN:
        BC_s_B_N<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(array, bc_s.sBN);
        break;
    }
  }

  /* TOP */
  if (dom[rank].t == MPI_PROC_NULL) {
    switch (bc_s.sT) {
      case DIRICHLET:
        BC_s_T_D<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(array, bc_s.sTD);
        break;
      case NEUMANN:
        BC_s_T_N<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(array, bc_s.sTN);
        break;
    }
  }
}

extern "C"
void cuda_scalar_part_BC(real *array)
{
  scalar_part_BC<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(array,
    _phase, _phase_shell, _parts, _s_parts);
}

extern "C"
void cuda_scalar_part_fill(void)
{
  scalar_part_fill<<<blocks.Gcc.num_kn, blocks.Gcc.dim_kn>>>(_s, _phase, _s_parts);
}

extern "C"
void cuda_scalar_solve(void)
{
  scalar_solve<<<blocks.Gcc.num_in, blocks.Gcc.dim_in>>>(_phase, _s0, _s,
    _s_conv, _s_diff, _s_conv0, _s_diff0, _u0, _v0, _w0, s_D, dt, dt0);
}

extern "C"
void cuda_scalar_partial_sum_i(void)
{
  //printf("N%d >> Communicating partial sums in i (nparts %d)\n", rank, nparts);
  /* Outline of communication of partial sums for Lebedev integration
   * 1) Finish local Lebedev integration in lebedev_quad<<<>>>. For a given
   *    scalar product, the partial sum for the jth coefficient of the nth
   *    particle is stored in: _int_someint[0 + NNODES*j + nparts*NNODES*n]
   * 2) All particles at the outermost two bin planes need their sums
   *    accumulated (e.g., (j,k) planes at _bins.Gcc.{_isb->_is,_ie->_ieb})
   * 3) Bin the particles using i indexing (find _bin_{start,end,count})
   * 4) Reduce _bin_count at _isb:_is, _ie:_ieb to find nparts_send_{e,w}
   * 5) Communicate nparts_send_{e,w} with adjacent subdomains to find
   *    nparts_recv_{w,e}
   * 6) Excl. prefix scan _bin_count over the _isb:_is, _ie:_ieb planes to find
   *    destination index for particle data packed into sending aray
   * 7) Allocate send array, int_send_{e,w} * 6 * sizeof(real). 6 comes from
   *    the number of integrals
   * 8) Allocate recv array, int_recv_{e,w} * 6 * sizeof(real).
   * 9) Communicate int_send_{e,w} to int_recv_{e,w}
   * 10)  Excl. prefix scan _bin_count over _isb:_is, _ie:_ieb planes to find unpacking
   *      incides - this already exists from earlier
   * 11)  Unpack and accumulate
   * 12)  Repeat for j, k
   */

  /* Initialize execution config */
  // Thread over east/west faces
  int ty = bins.Gcc.jnb * (bins.Gcc.jnb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.jnb >= MAX_THREADS_DIM);
  int tz = bins.Gcc.knb * (bins.Gcc.knb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.knb >= MAX_THREADS_DIM);

  int by = (int) ceil((real) bins.Gcc.jnb / (real) ty);
  int bz = (int) ceil((real) bins.Gcc.knb / (real) tz);

  dim3 bin_num_inb(by, bz);
  dim3 bin_dim_inb(ty, tz);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b, s2b; // custom strides
  int offset;

  /* Allocate */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_e;
  int *_offset_w;
  checkCudaErrors(hipMalloc(&_offset_e, 2 * bins.Gcc.s2b_i * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_w, 2 * bins.Gcc.s2b_i * sizeof(int)));
  thrust::device_ptr<int> t_offset_e(_offset_e);
  thrust::device_ptr<int> t_offset_w(_offset_w);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_i<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_i<<<bin_num_inb, bin_dim_inb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send and packing offsets */
    s1b = bins.Gcc.jnb;
    s2b = s1b * bins.Gcc.knb;

    // East: _ie and _ieb planes
    if (dom[rank].e != MPI_PROC_NULL) {
      // _bin_count is indexed with i varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _ie plane
      offset = GFX_LOC(bins.Gcc._ie, 0, 0, s1b, s2b);
      nparts_send[EAST] = thrust::reduce(t_bin_count + offset,
                                         t_bin_count + offset + 2 * bins.Gcc.s2b_i,
                                         0., thrust::plus<int>());

      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[EAST] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_i, t_offset_e);
      } else {
        hipMemset(_offset_e, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
      }

    } else { // no parts to send
      nparts_send[EAST] = 0;
      hipMemset(_offset_e, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
    }

    // West: _isb and _is planes
    if (dom[rank].w != MPI_PROC_NULL) {
      offset = GFX_LOC(bins.Gcc._isb, 0, 0, s1b, s2b);
      nparts_send[WEST] = thrust::reduce(t_bin_count + offset,
                                         t_bin_count + offset + 2 * bins.Gcc.s2b_i,
                                         0., thrust::plus<int>());
      if (nparts_send[WEST] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_i, t_offset_w);
      } else {
        hipMemset(_offset_w, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
      }

    } else {
      nparts_send[WEST] = 0;
      hipMemset(_offset_w, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
    }
  } else { // nparts <= 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[EAST] = 0;
    nparts_send[WEST] = 0;
    hipMemset(_offset_e, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
    hipMemset(_offset_w, 0., 2 * bins.Gcc.s2b_i * sizeof(int));
  }

  // Sending and receiving is the same since the outer two bin planes are shared
  nparts_recv[EAST] = nparts_send[EAST];
  nparts_recv[WEST] = nparts_send[WEST];

  /* Send number of parts to east/west */
  //    origin                target
  // nparts_send[WEST] -> nparts_recv[EAST]
  // nparts_recv[WEST] <- nparts_send[EAST]
  //nparts_recv[WEST] = 0; // init
  //nparts_recv[EAST] = 0;
  //mpi_send_nparts_i();

  /* Allocate memory for send and recv partial sums */
  int npsums = SNSP * s_ncoeffs_max;  // 2 scalar products * ncoeffs
  // Indexing is, for example:
  //  _sum_send_e[coeff + ncoeffs_max*sp + ncoeffs_max*nsp*part_id]
  // where
  //  part_id = [0, nparts) and sp = [0, 2)
  //    0:  Ys_re     1:  Ys_im

  // See accompanying note at the same location in cuda_transfer_parts_i
  int send_alloc_e = nparts_send[EAST]*(nparts_send[EAST] > 0) + (nparts_send[EAST] == 0);
  int send_alloc_w = nparts_send[WEST]*(nparts_send[WEST] > 0) + (nparts_send[WEST] == 0);
  int recv_alloc_e = nparts_recv[EAST]*(nparts_recv[EAST] > 0) + (nparts_recv[EAST] == 0);
  int recv_alloc_w = nparts_recv[WEST]*(nparts_recv[WEST] > 0) + (nparts_recv[WEST] == 0);

  checkCudaErrors(hipMalloc(&_sum_send_e, send_alloc_e*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_send_w, send_alloc_w*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_e, recv_alloc_e*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_w, recv_alloc_w*npsums*sizeof(real)));

  /* Pack partial sums */
  if (nparts_send[EAST] > 0) {
    pack_s_sums_e<<<bin_num_inb, bin_dim_inb>>>(_sum_send_e, _offset_e,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_e, 0., send_alloc_e * npsums * sizeof(real));
  }

  if (nparts_send[WEST] > 0) {
    pack_s_sums_w<<<bin_num_inb, bin_dim_inb>>>(_sum_send_w, _offset_w,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_w, 0., send_alloc_w * npsums * sizeof(real));
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Communicate partial sums with MPI */
  mpi_send_s_psums_i();

  // Offsets are the same since they're over both ghost bins and edge bins
  /* Unpack and complete partial sums */
  if (nparts_recv[EAST] > 0) {
    unpack_s_sums_e<<<bin_num_inb, bin_dim_inb>>>(_sum_recv_e, _offset_e,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  if (nparts_recv[WEST] > 0) {
    unpack_s_sums_w<<<bin_num_inb, bin_dim_inb>>>(_sum_recv_w, _offset_w,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Free */
  hipFree(_sum_send_e);
  hipFree(_sum_send_w);
  hipFree(_sum_recv_e);
  hipFree(_sum_recv_w);
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_e);
  hipFree(_offset_w);
}

extern "C"
void cuda_scalar_partial_sum_j(void)
{
  //printf("N%d >> Communicating partial sums in j\n", rank);
  /* Initialize execution config */
  // Thread over north/south faces
  int tz = bins.Gcc.knb * (bins.Gcc.knb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.knb >= MAX_THREADS_DIM);
  int tx = bins.Gcc.inb * (bins.Gcc.inb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.inb >= MAX_THREADS_DIM);

  int bz = (int) ceil((real) bins.Gcc.knb / (real) tz);
  int bx = (int) ceil((real) bins.Gcc.inb / (real) tx);

  dim3 bin_num_jnb(bz, bx);
  dim3 bin_dim_jnb(tz, tx);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b, s2b; // custom strides
  int offset;

  /* Allocate */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_n;
  int *_offset_s;
  checkCudaErrors(hipMalloc(&_offset_n, 2 * bins.Gcc.s2b_j * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_s, 2 * bins.Gcc.s2b_j * sizeof(int)));
  thrust::device_ptr<int> t_offset_n(_offset_n);
  thrust::device_ptr<int> t_offset_s(_offset_s);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_j<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_j<<<bin_num_jnb, bin_dim_jnb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send and packing offsets */
    s1b = bins.Gcc.knb;
    s2b = s1b * bins.Gcc.inb;

    // North: _je and _jeb planes
    if (dom[rank].n != MPI_PROC_NULL) {
      // _bin_count is indexed with i varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _je plane
      offset = GFY_LOC(0, bins.Gcc._je, 0, s1b, s2b);
      nparts_send[NORTH] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + 2 * bins.Gcc.s2b_j,
                                          0., thrust::plus<int>());

      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[NORTH] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_j, t_offset_n);
      } else {
        hipMemset(_offset_n, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
      }

    } else { // no parts to send
      nparts_send[NORTH] = 0;
      hipMemset(_offset_n, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
    }

    // South: _jsb and _js planes
    if (dom[rank].s != MPI_PROC_NULL) {
      offset = GFY_LOC(0, bins.Gcc._jsb, 0, s1b, s2b);
      nparts_send[SOUTH] = thrust::reduce(t_bin_count + offset,
                                          t_bin_count + offset + 2 * bins.Gcc.s2b_j,
                                          0., thrust::plus<int>());
      if (nparts_send[SOUTH] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_j, t_offset_s);
      } else {
        hipMemset(_offset_s, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
      }

    } else {
      nparts_send[SOUTH] = 0;
      hipMemset(_offset_s, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
    }
  } else { // nparts == 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[NORTH] = 0;
    nparts_send[SOUTH] = 0;
    hipMemset(_offset_n, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
    hipMemset(_offset_s, 0., 2 * bins.Gcc.s2b_j * sizeof(int));
  }

  // Sending and receiving is the same since the outer two bin planes are shared
  nparts_recv[NORTH] = nparts_send[NORTH];
  nparts_recv[SOUTH] = nparts_send[SOUTH];

  /* Send number of parts to north/south */
  //    origin                target
  // nparts_send[SOUTH] -> nparts_recv[NORTH]
  // nparts_recv[SOUTH] <- nparts_send[NORTH]
  //nparts_recv[SOUTH] = 0; // init
  //nparts_recv[NORTH] = 0;
  //mpi_send_nparts_j();

  /* Allocate memory for send and recv partial sums */
  int npsums = SNSP * s_ncoeffs_max;  // 2 scalar products * ncoeffs
  // Indexing is, for example:
  //  _sum_send_n[coeff + ncoeffs_max*sp + ncoeffs_max*nsp*part_id]
  // where
  //  part_id = [0, nparts) and sp = [0, 2)
  //    0:  Ys_re     1:  Ys_im

  // See accompanying note at the same location in cuda_transfer_parts_i
  int send_alloc_n = nparts_send[NORTH]*(nparts_send[NORTH] > 0) + (nparts_send[NORTH] == 0);
  int send_alloc_s = nparts_send[SOUTH]*(nparts_send[SOUTH] > 0) + (nparts_send[SOUTH] == 0);
  int recv_alloc_n = nparts_recv[NORTH]*(nparts_recv[NORTH] > 0) + (nparts_recv[NORTH] == 0);
  int recv_alloc_s = nparts_recv[SOUTH]*(nparts_recv[SOUTH] > 0) + (nparts_recv[SOUTH] == 0);

  checkCudaErrors(hipMalloc(&_sum_send_n, send_alloc_n*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_send_s, send_alloc_s*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_n, recv_alloc_n*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_s, recv_alloc_s*npsums*sizeof(real)));

  /* Pack partial sums */
  if (nparts_send[NORTH] > 0) {
    pack_s_sums_n<<<bin_num_jnb, bin_dim_jnb>>>(_sum_send_n, _offset_n,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_n, 0., send_alloc_n * npsums * sizeof(real));
  }

  if (nparts_send[SOUTH] > 0) {
    pack_s_sums_s<<<bin_num_jnb, bin_dim_jnb>>>(_sum_send_s, _offset_s,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_s, 0., send_alloc_s * npsums * sizeof(real));
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Communicate partial sums with MPI */
  mpi_send_s_psums_j();

  // Offsets are the same since they're over both ghost bins and edge bins
  /* Unpack and complete partial sums */
  if (nparts_recv[NORTH] > 0) {
    unpack_s_sums_n<<<bin_num_jnb, bin_dim_jnb>>>(_sum_recv_n, _offset_n,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  if (nparts_recv[SOUTH] > 0) {
    unpack_s_sums_s<<<bin_num_jnb, bin_dim_jnb>>>(_sum_recv_s, _offset_s,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Free */
  hipFree(_sum_send_n);
  hipFree(_sum_send_s);
  hipFree(_sum_recv_n);
  hipFree(_sum_recv_s);
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_n);
  hipFree(_offset_s);
}

extern "C"
void cuda_scalar_partial_sum_k(void)
{
  //printf("N%d >> Communicating partial sums in k\n", rank);
  /* Initialize execution config */
  // Thread over top/bottom faces
  int tx = bins.Gcc.inb * (bins.Gcc.inb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.inb >= MAX_THREADS_DIM);
  int ty = bins.Gcc.jnb * (bins.Gcc.jnb < MAX_THREADS_DIM)
       + MAX_THREADS_DIM * (bins.Gcc.jnb >= MAX_THREADS_DIM);

  int bx = (int) ceil((real) bins.Gcc.inb / (real) tx);
  int by = (int) ceil((real) bins.Gcc.jnb / (real) ty);

  dim3 bin_num_knb(bx, by);
  dim3 bin_dim_knb(tx, ty);

  // Thread over nparts
  int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
  int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

  dim3 dim_nparts(t_nparts);
  dim3 num_nparts(b_nparts);

  /* Declare things we might need */
  int s1b, s2b; // custom strides
  int offset;

  /* Allocate */
  checkCudaErrors(hipMalloc(&_part_ind, nparts * sizeof(int)));
  checkCudaErrors(hipMalloc(&_part_bin, nparts * sizeof(int)));
  thrust::device_ptr<int> t_part_ind(_part_ind);
  thrust::device_ptr<int> t_part_bin(_part_bin);

  int *_offset_t;
  int *_offset_b;
  checkCudaErrors(hipMalloc(&_offset_t, 2 * bins.Gcc.s2b_k * sizeof(int)));
  checkCudaErrors(hipMalloc(&_offset_b, 2 * bins.Gcc.s2b_k * sizeof(int)));
  thrust::device_ptr<int> t_offset_t(_offset_t);
  thrust::device_ptr<int> t_offset_b(_offset_b);

  checkCudaErrors(hipMemset(_bin_start, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_end, -1, bins.Gcc.s3b * sizeof(int)));
  checkCudaErrors(hipMemset(_bin_count, 0, bins.Gcc.s3b * sizeof(int)));
  thrust::device_ptr<int> t_bin_count(_bin_count);

  if (nparts > 0) {
    /* Find each particle's bin */
    bin_fill_k<<<num_nparts, dim_nparts>>>(_part_ind, _part_bin, _parts, nparts,
      _DOM);

    /* Sort _part_ind by _part_bin (sort key by value) */
    if (nparts > 1) {
      thrust::sort_by_key(t_part_bin, t_part_bin + nparts, t_part_ind);
    }

    /* Find start and ending index of each bin */
    int smem_size = (nparts + 1) * sizeof(int);
    find_bin_start_end<<<b_nparts, t_nparts, smem_size>>>(_bin_start, _bin_end,
      _part_bin, nparts);

    /* Find number of particles in each bin */
    count_bin_parts_k<<<bin_num_knb, bin_dim_knb>>>(_bin_start, _bin_end,
      _bin_count);

    /* Find number of particles to send and packing offsets */
    s1b = bins.Gcc.inb;
    s2b = s1b * bins.Gcc.jnb;

    // North: _ke and _keb planes
    if (dom[rank].t != MPI_PROC_NULL) {
      // _bin_count is indexed with k varying slowest -- can do a reduction
      // directly from _bin_count, given the offset of the start of the _ke plane
      offset = GFZ_LOC(0, 0, bins.Gcc._ke, s1b, s2b);
      nparts_send[TOP] = thrust::reduce(t_bin_count + offset,
                                        t_bin_count + offset + 2 * bins.Gcc.s2b_k,
                                        0., thrust::plus<int>());

      /* Determine packing offsets with an excl prefix scan */
      if (nparts_send[TOP] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_k, t_offset_t);
      } else {
        hipMemset(_offset_t, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
      }

    } else { // no parts to send
      nparts_send[TOP] = 0;
      hipMemset(_offset_t, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
    }

    // South: _ksb and _ks planes
    if (dom[rank].b != MPI_PROC_NULL) {
      offset = GFZ_LOC(0, 0, bins.Gcc._ksb, s1b, s2b);
      nparts_send[BOTTOM] = thrust::reduce(t_bin_count + offset,
                                           t_bin_count + offset + 2 * bins.Gcc.s2b_k,
                                           0., thrust::plus<int>());
      if (nparts_send[BOTTOM] > 0) {
        thrust::exclusive_scan(t_bin_count + offset,
                               t_bin_count + offset + 2 * bins.Gcc.s2b_k, t_offset_b);
      } else {
        hipMemset(_offset_b, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
      }

    } else {
      nparts_send[BOTTOM] = 0;
      hipMemset(_offset_b, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
    }
  } else { // nparts = 0
    checkCudaErrors(hipMemset(_part_ind, -1, nparts * sizeof(int)));
    checkCudaErrors(hipMemset(_part_bin, -1, nparts * sizeof(int)));
    nparts_send[TOP] = 0;
    nparts_send[BOTTOM] = 0;
    hipMemset(_offset_t, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
    hipMemset(_offset_b, 0., 2 * bins.Gcc.s2b_k * sizeof(int));
  }

  // Sending and receiving is the same since the outer two bin planes are shared
  nparts_recv[TOP] = nparts_send[TOP];
  nparts_recv[BOTTOM] = nparts_send[BOTTOM];

  /* Send number of parts to top/bottom */
  //    origin                target
  // nparts_send[BOTTOM] -> nparts_recv[TOP]
  // nparts_recv[BOTTOM] <- nparts_send[TOP]
  //nparts_recv[BOTTOM] = 0; // init
  //nparts_recv[TOP] = 0;
  //mpi_send_nparts_k();

  /* Allocate memory for send and recv partial sums */
  int npsums = SNSP * s_ncoeffs_max;  // 2 scalar products * ncoeffs
  // Indexing is, for example:
  //  _sum_send_t[coeff + ncoeffs_max*sp + ncoeffs_max*nsp*part_id]
  // where
  //  part_id = [0, nparts) and sp = [0, 2)
  //    0:  Ys_re     1:  Ys_im

  int send_alloc_t = nparts_send[TOP]*(nparts_send[TOP] > 0) + (nparts_send[TOP] == 0);
  int send_alloc_b = nparts_send[BOTTOM]*(nparts_send[BOTTOM] > 0) + (nparts_send[BOTTOM] == 0);
  int recv_alloc_t = nparts_recv[TOP]*(nparts_recv[TOP] > 0) + (nparts_recv[TOP] == 0);
  int recv_alloc_b = nparts_recv[BOTTOM]*(nparts_recv[BOTTOM] > 0) + (nparts_recv[BOTTOM] == 0);

  checkCudaErrors(hipMalloc(&_sum_send_t, send_alloc_t*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_send_b, send_alloc_b*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_t, recv_alloc_t*npsums*sizeof(real)));
  checkCudaErrors(hipMalloc(&_sum_recv_b, recv_alloc_b*npsums*sizeof(real)));

  /* Pack partial sums */
  if (nparts_send[TOP] > 0) {
    pack_s_sums_t<<<bin_num_knb, bin_dim_knb>>>(_sum_send_t, _offset_t,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_t, 0., send_alloc_t * npsums * sizeof(real));
  }

  if (nparts_send[BOTTOM] > 0) {
    pack_s_sums_b<<<bin_num_knb, bin_dim_knb>>>(_sum_send_b, _offset_b,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  } else {
    //hipMemset(_sum_send_b, 0., send_alloc_b * npsums * sizeof(real));
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Communicate partial sums with MPI */
  mpi_send_s_psums_k();

  // Offsets are the same since they're over both ghost bins and edge bins
  /* Unpack and complete partial sums */
  if (nparts_recv[TOP] > 0) {
    unpack_s_sums_t<<<bin_num_knb, bin_dim_knb>>>(_sum_recv_t, _offset_t,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  if (nparts_recv[BOTTOM] > 0) {
    unpack_s_sums_b<<<bin_num_knb, bin_dim_knb>>>(_sum_recv_b, _offset_b,
      _bin_start, _bin_count, _part_ind, s_ncoeffs_max,
      _int_Ys_re, _int_Ys_im);
  }
  hipDeviceSynchronize();  // ensure packing is complete

  /* Free */
  hipFree(_sum_send_t);
  hipFree(_sum_send_b);
  hipFree(_sum_recv_t);
  hipFree(_sum_recv_b);
  hipFree(_part_ind);
  hipFree(_part_bin);
  hipFree(_offset_t);
  hipFree(_offset_b);
}

extern "C"
void cuda_scalar_lamb(void)
{
  /* CUDA exec config */
  dim3 num_parts(nparts); // nparts blocks with nnodes threads each
  dim3 dim_nodes(NNODES);
  dim3 num_partcoeff(nparts, s_ncoeffs_max);
  dim3 dim_coeff(s_ncoeffs_max);

  //printf("N%d >> Determining Lamb's coefficients (nparts = %d)\n", rank, nparts);
  if (nparts > 0) {
    /* Temp storage for field variables at quadrature nodes */
    real *_ss;    // scalar
    checkCudaErrors(hipMalloc(&_ss, NNODES * nparts * sizeof(real)));

    /* Interpolate field varaibles to quadrature nodes */
    scalar_check_nodes<<<num_parts, dim_nodes>>>(_parts, _s_parts, _bc_s, _DOM);
    scalar_interpolate_nodes<<<num_parts, dim_nodes>>>(_s, _ss,
      _parts, _s_parts, _bc_s);

    /* Create scalar product storage using max particle coefficient size */
    int sp_size = nparts * NNODES * s_ncoeffs_max;
    checkCudaErrors(hipMalloc(&_int_Ys_re, sp_size * sizeof(real)));
    checkCudaErrors(hipMalloc(&_int_Ys_im, sp_size * sizeof(real)));

    /* Perform partial sums of lebedev quadrature */
    scalar_lebedev_quadrature<<<num_partcoeff, dim_nodes>>>(_parts,
      _s_parts, s_ncoeffs_max, _ss, _int_Ys_re, _int_Ys_im);

    checkCudaErrors(hipFree(_ss));
  }

  /* Accumulate partial sums (all procs need to be involved) */
  cuda_scalar_partial_sum_i();  // 2a) Calculate partial sums over x face
  cuda_scalar_partial_sum_j();  // 2b) Calculate partial sums over y face
  cuda_scalar_partial_sum_k();  // 2c) Calculate partial sums over z face

  if (nparts > 0) {
    /* Compute lambs coefficients from partial sums */
    scalar_compute_coeffs<<<num_parts, dim_coeff>>>(_parts, _s_parts,
      s_ncoeffs_max, nparts, _int_Ys_re, _int_Ys_im);

    /* Free */
    checkCudaErrors(hipFree(_int_Ys_re));
    checkCudaErrors(hipFree(_int_Ys_im));
  }
}

extern "C"
real cuda_scalar_lamb_err(void)
{
  //printf("N%d >> Determining Lamb's error\n", rank);
  real error = DBL_MIN;
  if (nparts > 0) {
    // create a place to store errors
    real *_part_errors;
    hipMalloc((void**) &_part_errors, nparts*sizeof(real));
    
    // sort the coefficients and calculate errors along the way
    dim3 numBlocks(nparts);
    dim3 dimBlocks(s_ncoeffs_max);

    scalar_compute_error<<<numBlocks, dimBlocks>>>(lamb_cut_scalar,
     s_ncoeffs_max, nparts, _s_parts, _part_errors);

    // find maximum error of all particles
    thrust::device_ptr<real> t_part_errors(_part_errors);
    error = thrust::reduce(t_part_errors,
                           t_part_errors + nparts,
                           0., thrust::maximum<real>());

    // clean up
    hipFree(_part_errors);

    // store copy of coefficients for future calculation
    scalar_store_coeffs<<<numBlocks, dimBlocks>>>(_s_parts, nparts, s_ncoeffs_max);
  }

  // MPI reduce to find max error
  MPI_Allreduce(MPI_IN_PLACE, &error, 1, mpi_real, MPI_MAX, MPI_COMM_WORLD);
  return error;
}

extern "C"
void cuda_store_s(void)
{
  checkCudaErrors(hipMemcpy(_s0, _s, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(_s_conv0, _s_conv, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(_s_diff0, _s_diff, dom[rank].Gcc.s3b * sizeof(real),
    hipMemcpyDeviceToDevice));
}

void cuda_scalar_update_part(void)
{
  if(nparts > 0) {
    int t_nparts = nparts * (nparts < MAX_THREADS_1D)
                  + MAX_THREADS_1D * (nparts >= MAX_THREADS_1D);
    int b_nparts = (int) ceil((real) nparts / (real) t_nparts);

    dim3 dim_nparts(t_nparts);
    dim3 num_nparts(b_nparts);
    update_part_scalar<<<num_nparts, dim_nparts>>>(_parts, _s_parts, ttime, dt, s_k);
  }
}
