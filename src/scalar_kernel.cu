#include "hip/hip_runtime.h"
#include "cuda_physalis.h"
#include "cuda_scalar.h"

__global__ void pack_s_parts_e(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    cbin = GFX_LOC(_bins.Gcc._ie, tj, tk, s1b, s2b);
    c2b = tj + tk * _bins.Gcc.jnb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void pack_s_parts_w(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    cbin = GFX_LOC(_bins.Gcc._is, tj, tk, s1b, s2b);
    c2b = tj + tk * _bins.Gcc.jnb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void pack_s_parts_n(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    cbin = GFY_LOC(ti, _bins.Gcc._je, tk, s1b, s2b);
    c2b = tk + ti * _bins.Gcc.knb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void pack_s_parts_s(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    cbin = GFY_LOC(ti, _bins.Gcc._js, tk, s1b, s2b);
    c2b = tk + ti * _bins.Gcc.knb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void pack_s_parts_t(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    cbin = GFZ_LOC(ti, tj, _bins.Gcc._ke, s1b, s2b);
    c2b = ti + tj * _bins.Gcc.inb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void pack_s_parts_b(part_struct_scalar *send_parts, part_struct_scalar *parts,
  int *offset, int *bin_start, int *bin_count, int *part_ind)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    cbin = GFZ_LOC(ti, tj, _bins.Gcc._ks, s1b, s2b);
    c2b = ti + tj * _bins.Gcc.inb;

    // Loop through each bin's particles and add to send_parts
    // Each bin is offset by offset[cbin] (from excl. prefix scan)
    // Each particle is then offset from that
    for (int i = 0; i < bin_count[cbin]; i++) {
      pp = part_ind[bin_start[cbin] + i];
      send_parts[offset[c2b] + i] = parts[pp];
    }
  }
}

__global__ void copy_central_bin_s_parts_i(part_struct_scalar *tmp_parts,
  part_struct_scalar *parts, int *bin_start, int *bin_count, int *part_ind,
  int *offset)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;             // bin index
  int pp;               // particle index
  int dest;             // destination in tmp_parts

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    // Loop over i-planes
    for (int i = _bins.Gcc._is; i <= _bins.Gcc._ie; i++) {
      cbin = GFX_LOC(i, tj, tk, s1b, s2b);


      for (int n = 0; n < bin_count[cbin]; n++) {
        pp = part_ind[bin_start[cbin] + n];
        dest = offset[cbin] + n;

        tmp_parts[dest] = parts[pp];
      }
    }
  }
}

__global__ void copy_central_bin_s_parts_j(part_struct_scalar *tmp_parts,
  part_struct_scalar *parts, int *bin_start, int *bin_count, int *part_ind,
  int *offset)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;             // bin index
  int pp;               // particle index
  int dest;             // destination in tmp_parts

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    // Loop over j-planes
    for (int j = _bins.Gcc._js; j <= _bins.Gcc._je; j++) {
      cbin = GFY_LOC(ti, j, tk, s1b, s2b);

      for (int n = 0; n < bin_count[cbin]; n++) {
        pp = part_ind[bin_start[cbin] + n];
        dest = offset[cbin] + n;

        tmp_parts[dest] = parts[pp];
      }
    }
  }
}

__global__ void copy_central_bin_s_parts_k(part_struct_scalar *tmp_parts,
  part_struct_scalar *parts, int *bin_start, int *bin_count, int *part_ind,
  int *offset)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;             // bin index
  int pp;               // particle index
  int dest;             // destination in tmp_parts

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    // Loop over j-planes
    for (int k = _bins.Gcc._ks; k <= _bins.Gcc._ke; k++) {
      cbin = GFZ_LOC(ti, tj, k, s1b, s2b);

      for (int n = 0; n < bin_count[cbin]; n++) {
        pp = part_ind[bin_start[cbin] + n];
        dest = offset[cbin] + n;

        tmp_parts[dest] = parts[pp];
      }
    }
  }
}

__global__ void copy_ghost_bin_s_parts(part_struct_scalar *tmp_parts,
  part_struct_scalar *recv_parts, int nparts_recv, int offset, int plane, dom_struct *DOM)
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x; // particle index
  int dest;

  if (pp < nparts_recv) {
    dest = offset + pp;
    tmp_parts[dest] = recv_parts[pp];
  }
}

__global__ void BC_s_W_D(real *array, real bc_s)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if((tj < _dom.Gcc.jn) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(_dom.Gcc._isb, tj + 1, tk + 1, s1b, s2b)] = bc_s;
}

__global__ void BC_s_W_N(real *array, real bc_s)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if((tj < _dom.Gcc.jn) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(_dom.Gcc._isb, tj + 1, tk + 1, s1b, s2b)] = 
      array[GCC_LOC(_dom.Gcc._is, tj + 1, tk + 1, s1b, s2b)] - bc_s*_dom.dx;
}

__global__ void BC_s_E_D(real *array, real bc_s)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((tj < _dom.Gcc.jn) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(_dom.Gcc._ieb, tj + 1, tk + 1, s1b, s2b)] = bc_s;
}

__global__ void BC_s_E_N(real *array, real bc_s)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((tj < _dom.Gcc.jn) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(_dom.Gcc._ieb, tj + 1, tk + 1, s1b, s2b)] = 
      array[GCC_LOC(_dom.Gcc._ie, tj + 1, tk + 1, s1b, s2b)] + _dom.dx*bc_s;
}

__global__ void BC_s_N_D(real *array, real bc_s)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(ti + 1, _dom.Gcc._jeb, tk + 1, s1b, s2b)] = bc_s;
}

__global__ void BC_s_N_N(real *array, real bc_s)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(ti + 1, _dom.Gcc._jeb, tk + 1, s1b, s2b)] = 
      array[GCC_LOC(ti + 1, _dom.Gcc._je, tk + 1, s1b, s2b)] + _dom.dy*bc_s;
}

__global__ void BC_s_S_D(real *array, real bc_s)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(ti + 1, _dom.Gcc._jsb, tk + 1, s1b, s2b)] = bc_s;
}

__global__ void BC_s_S_N(real *array, real bc_s)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tk < _dom.Gcc.kn))
    array[GCC_LOC(ti + 1, _dom.Gcc._jsb, tk + 1, s1b, s2b)] = 
      array[GCC_LOC(ti + 1, _dom.Gcc._js, tk + 1, s1b, s2b)] - _dom.dy*bc_s;
}

__global__ void BC_s_B_D(real *array, real bc_s)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tj < _dom.Gcc.jn))
    array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._ksb, s1b, s2b)] = bc_s;
}

__global__ void BC_s_B_N(real *array, real bc_s)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tj < _dom.Gcc.jn))
    array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._ksb, s1b, s2b)] = 
      array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._ks, s1b, s2b)] - _dom.dz*bc_s;
}

__global__ void BC_s_T_D(real *array, real bc_s)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tj < _dom.Gcc.jn))
    array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._keb, s1b, s2b)] = bc_s;
}

__global__ void BC_s_T_N(real *array, real bc_s)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  int s1b = _dom.Gcc.s1b;
  int s2b = _dom.Gcc.s2b;

  if ((ti < _dom.Gcc.in) && (tj < _dom.Gcc.jn))
    array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._keb, s1b, s2b)] = 
      array[GCC_LOC(ti + 1, tj + 1, _dom.Gcc._ke, s1b, s2b)] + _dom.dz*bc_s;
}

__global__ void forcing_boussinesq_x(real alpha, real gx, real s_init, real *s, real *fx)
{
  int i, C0, C1;
  int tj = blockIdx.x * blockDim.x + threadIdx.x;
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  if(tj < _dom.Gfx.jnb && tk < _dom.Gfx.knb) {
    for(i = _dom.Gfx._isb + 1; i <= _dom.Gfx._ieb - 1; i++) {
	  C0 = GCC_LOC(i-1, tj, tk, _dom.Gcc.s1b, _dom.Gcc.s2b);
	  C1 = GCC_LOC(i  , tj, tk, _dom.Gcc.s1b, _dom.Gcc.s2b);
      fx[GFX_LOC(i, tj, tk, _dom.Gfx.s1b, _dom.Gfx.s2b)]
        += - gx * alpha * (0.5*(s[C0]+s[C1]) - s_init);
    }
  }
}

__global__ void forcing_boussinesq_y(real alpha, real gy, real s_init, real *s, real *fy)
{
  int j, C0, C1;
  int tk = blockIdx.x * blockDim.x + threadIdx.x;
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  if(tk < _dom.Gfy.knb && ti < _dom.Gfy.inb) {
    for(j = _dom.Gfy._jsb + 1; j <= _dom.Gfy._jeb - 1; j++) {
	  C0 = GCC_LOC(ti, j-1, tk, _dom.Gcc.s1b, _dom.Gcc.s2b);
	  C1 = GCC_LOC(ti, j  , tk, _dom.Gcc.s1b, _dom.Gcc.s2b);
      fy[GFY_LOC(ti, j, tk, _dom.Gfy.s1b, _dom.Gfy.s2b)]
        += - gy * alpha * (0.5*(s[C0]+s[C1]) - s_init);
    }
  }
}

__global__ void forcing_boussinesq_z(real alpha, real gz, real s_init, real *s, real *fz)
{
  int k, C0, C1;
  int ti = blockIdx.x * blockDim.x + threadIdx.x;
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  if(ti < _dom.Gfz.inb && tj < _dom.Gfz.jnb) {
    for(k = _dom.Gfz._ksb + 1; k <= _dom.Gfz._keb - 1; k++) {
	  C0 = GCC_LOC(ti, tj, k-1, _dom.Gcc.s1b, _dom.Gcc.s2b);
	  C1 = GCC_LOC(ti, tj, k  , _dom.Gcc.s1b, _dom.Gcc.s2b);
      fz[GFZ_LOC(ti, tj, k, _dom.Gfz.s1b, _dom.Gfz.s2b)]
        += - gz * alpha * (0.5*(s[C0]+s[C1]) - s_init);
    }
  }
}

__global__ void copy_subdom_parts_with_scalar(part_struct *tmp_parts, part_struct *parts,
  part_struct_scalar *tmp_s_parts, part_struct_scalar *s_parts,
  int *bin_start, int *bin_count, int *part_ind, int *bin_offset)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;             // bin index
  int pp;               // particle index
  int dest;             // destination in tmp_parts

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.in && tj < _bins.Gcc.jn) {
    for (int k = _bins.Gcc._ks; k <= _bins.Gcc._ke; k++) {
      cbin = GFZ_LOC(ti + 1, tj + 1, k, s1b, s2b);

      for (int n = 0; n < bin_count[cbin]; n++) {
        pp = part_ind[bin_start[cbin] + n];
        dest = bin_offset[cbin] + n;

        tmp_parts[dest] = parts[pp];
        tmp_s_parts[dest] = s_parts[pp];
      }
    }
  }
}

__global__ void scalar_solve(int *phase, real *s0, real *s,
  real *conv, real *diff, real *conv0, real *diff0,
  real *u0, real *v0, real *w0, real D, real dt, real dt0)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x + DOM_BUF;
  int tk = blockIdx.y * blockDim.y + threadIdx.y + DOM_BUF;

  // working constants
  real ab0 = 0.5 * dt / dt0;
  real ab = 1. + ab0;
  int C, Cx0, Cx1, Cy0, Cy1, Cz0, Cz1;
  int fx0, fx1, fy0, fy1, fz0, fz1;
  real conv_x, conv_y, conv_z;
  real diff_x, diff_y, diff_z;

  // loop over x-plane
  if(tj <= _dom.Gcc._je && tk <= _dom.Gcc._ke) {
    for(int i = _dom.Gcc._is; i <= _dom.Gcc._ie; i++) {
      C   = GCC_LOC(i,   tj,   tk,   _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cx0 = GCC_LOC(i-1, tj,   tk,   _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cx1 = GCC_LOC(i+1, tj,   tk,   _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cy0 = GCC_LOC(i,   tj-1, tk,   _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cy1 = GCC_LOC(i,   tj+1, tk,   _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cz0 = GCC_LOC(i,   tj,   tk-1, _dom.Gcc.s1b, _dom.Gcc.s2b);
      Cz1 = GCC_LOC(i,   tj,   tk+1, _dom.Gcc.s1b, _dom.Gcc.s2b);
      fx0 = GFX_LOC(i,   tj,   tk,   _dom.Gfx.s1b, _dom.Gfx.s2b);
      fx1 = GFX_LOC(i+1, tj,   tk,   _dom.Gfx.s1b, _dom.Gfx.s2b);
      fy0 = GFY_LOC(i,   tj,   tk,   _dom.Gfy.s1b, _dom.Gfy.s2b);
      fy1 = GFY_LOC(i,   tj+1, tk,   _dom.Gfy.s1b, _dom.Gfy.s2b);
      fz0 = GFZ_LOC(i,   tj,   tk,   _dom.Gfz.s1b, _dom.Gfz.s2b);
      fz1 = GFZ_LOC(i,   tj,   tk+1, _dom.Gfz.s1b, _dom.Gfz.s2b);

      // calculate the convection term
      conv_x = u0[fx1] * 0.5 * (s0[Cx1] + s0[C]) - u0[fx0] * 0.5 * (s0[C] + s0[Cx0]);
      conv_x = conv_x / _dom.dx;
      conv_y = v0[fy1] * 0.5 * (s0[Cy1] + s0[C]) - v0[fy0] * 0.5 * (s0[C] + s0[Cy0]);
      conv_y = conv_y / _dom.dy;
      conv_z = w0[fz1] * 0.5 * (s0[Cz1] + s0[C]) - w0[fz0] * 0.5 * (s0[C] + s0[Cz0]);
      conv_z = conv_z / _dom.dz;  
      conv[C] = conv_x + conv_y + conv_z;

      // calculate the diffusion term
      diff_x = D * (s0[Cx0] - 2.*s0[C] + s0[Cx1]) / _dom.dx / _dom.dx;
      diff_y = D * (s0[Cy0] - 2.*s0[C] + s0[Cy1]) / _dom.dy / _dom.dy;
      diff_z = D * (s0[Cz0] - 2.*s0[C] + s0[Cz1]) / _dom.dz / _dom.dz;
      diff[C] = diff_x + diff_y + diff_z;

      // Adams-Bashforth
      if(phase[C] == -1) {
        if(dt0 > 0) {
          s[C] = s0[C] + dt * (ab * diff[C] - ab0 * diff0[C] - (ab * conv[C] - ab0 * conv0[C]));
        } else {
          s[C] = s0[C] + dt * (diff[C] - conv[C]);
        }
      }
    }
  }
}

__global__ void scalar_check_nodes(part_struct *parts,
  part_struct_scalar *s_parts, BC_s *bc_s, dom_struct *DOM)
{
  int node = threadIdx.x;
  int part = blockIdx.x;

  /* Convert node (r, theta, phi) to (x, y, z) */
  real xp, yp, zp;  // Cartesian radial vector
  real x, y, z;     // Cartesian location of node
  rtp2xyz(s_parts[part].rs, _node_t[node], _node_p[node], &xp, &yp, &zp);

  /* shift from particle center */
  x = xp + parts[part].x;
  y = yp + parts[part].y;
  z = zp + parts[part].z;

  // start off with all -1's
  parts[part].nodes[node] = -1;

  /* check if the node is interfered with by a wall */
  // compute distance between node and walls
  // set equal to some number to identify which wall is interfering

  // We use <= for E,N,T and > for W,S,B -- allows us to do [start,end) on all 
  // subdomains regardless of bc
  parts[part].nodes[node] += (WEST_WALL + 1) *    // set equal to WEST_WALL...
              (x - _dom.xs < 0) *                 // if outside domain &
              (_dom.I == DOM->Is) *                // if edge domain & DIRICHLET
              (bc_s->sW == DIRICHLET)*
              (parts[part].nodes[node] == -1);

  parts[part].nodes[node] += (EAST_WALL + 1) * 
              (x - _dom.xe >= 0) *
              (_dom.I == DOM->Ie) *
              (bc_s->sE == DIRICHLET)*
              (parts[part].nodes[node] == -1);

  parts[part].nodes[node] += (SOUTH_WALL + 1) *
              (y - _dom.ys < 0) *
              (_dom.J == DOM->Js) *
              (bc_s->sS == DIRICHLET)*
              (parts[part].nodes[node] == -1);

  parts[part].nodes[node] += (NORTH_WALL + 1) *
              (y - _dom.ye >= 0) *
              (_dom.J == DOM->Je) *
              (bc_s->sN == DIRICHLET)*
              (parts[part].nodes[node] == -1);

  parts[part].nodes[node] += (BOTTOM_WALL + 1) *
              (z - _dom.zs < 0) *
              (_dom.K == DOM->Ks) *
              (bc_s->sB == DIRICHLET)*
              (parts[part].nodes[node] == -1);

  parts[part].nodes[node] += (TOP_WALL + 1) *
              (z - _dom.ze >= 0) *
              (_dom.K == DOM->Ke) *
              (bc_s->sT == DIRICHLET)*
              (parts[part].nodes[node] == -1);
}

__global__ void scalar_interpolate_nodes(real *s, real *ss,
  part_struct *parts, part_struct_scalar *s_parts, BC_s *bc_s)
{
  int node = threadIdx.x;
  int part = blockIdx.x;

  real ddx = 1. / _dom.dx;
  real ddy = 1. / _dom.dy;
  real ddz = 1. / _dom.dz;

  real sswall;

  int i, j, k;  // index of cells containing node
  int oob;      // out of bounds indicator, 1 if out of bounds else 0
  int C, Ce, Cw, Cn, Cs, Ct, Cb;  // cell indices
  real xx, yy, zz;  // Cartesian location of s

  // convert node (r, theta, phi) to (x, y, z)
  real xp, yp, zp;  // Cartesian radial vector
  real x, y, z;     // Cartesian location of node
  rtp2xyz(s_parts[part].rs, _node_t[node], _node_p[node], &xp, &yp, &zp);

  // shift from particle center
  x = xp + parts[part].x;
  y = yp + parts[part].y;
  z = zp + parts[part].z;

  /* Find index of cell containing node. */
  // Do this in GLOBAL coordinates so that magnitude of floating point error is
  // the same on each subdomain.
  real arg_x = (x - (_dom.xs - _dom.dx)) * ddx + _dom.Gcc.isb;
  real arg_y = (y - (_dom.ys - _dom.dy)) * ddy + _dom.Gcc.jsb;
  real arg_z = (z - (_dom.zs - _dom.dz)) * ddz + _dom.Gcc.ksb;

  /* Deal with floating point errors in position so we don't lose nodes */
  // Similar to bin_fill_{i,j,k}. If floor != round and round is "close enough"
  // to the nearest integer, use round instead. this ensures that all nodes are
  // accounted for between subdomains
  // Using global indices makes sure that the floating point representation
  // error is the same for each subdomain, since the magnitude of the index will
  // be the similar/the same.

  i = floor(arg_x);
  j = floor(arg_y);
  k = floor(arg_z);

  int round_x = lrint(arg_x);
  int round_y = lrint(arg_y);
  int round_z = lrint(arg_z);

  // Better way to do this? no if-statement... abs?
  if ((round_x != i) && (abs(round_x - arg_x) <= DBL_EPSILON)) {
    i = round_x;
  }
  if ((round_y != j) && (abs(round_y - arg_y) <= DBL_EPSILON)) {
    j = round_y;
  }
  if ((round_z != k) && (abs(round_z - arg_z) <= DBL_EPSILON)) {
    k = round_z;
  }

  // Convert back to LOCAL coodrinates
  i -= _dom.Gcc.isb;
  j -= _dom.Gcc.jsb;
  k -= _dom.Gcc.ksb;

  /* Interpolate Scalar */
  // Find if out-of-bounds -- 1 if oob, 0 if in bounds
  oob = i < _dom.Gcc._is || i >= _dom.Gcc._ie ||
        j < _dom.Gcc._js || j >= _dom.Gcc._je ||
        k < _dom.Gcc._ks || k >= _dom.Gcc._ke;

  // Correct indices so we don't have out-of-bounds reads
  // If out out bounds, we'll read good info but trash the results
  i += (_dom.Gcc._is - i) * (i < _dom.Gcc._is);
  j += (_dom.Gcc._js - j) * (j < _dom.Gcc._js);
  k += (_dom.Gcc._ks - k) * (k < _dom.Gcc._is);
  i += (_dom.Gcc._ie - i) * (i > _dom.Gcc._ie);
  j += (_dom.Gcc._je - j) * (j > _dom.Gcc._je);
  k += (_dom.Gcc._ke - k) * (k > _dom.Gcc._ke);

  // Cell-centered indices
  C = GCC_LOC(i, j, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Ce = GCC_LOC(i + 1, j, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Cw = GCC_LOC(i - 1, j, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Cn = GCC_LOC(i, j + 1, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Cs = GCC_LOC(i, j - 1, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Ct = GCC_LOC(i, j, k + 1, _dom.Gcc.s1b, _dom.Gcc.s2b);
  Cb = GCC_LOC(i, j, k - 1, _dom.Gcc.s1b, _dom.Gcc.s2b);

  // Cartesian location of center of cell
  xx = (i - 0.5) * _dom.dx + _dom.xs;
  yy = (j - 0.5) * _dom.dy + _dom.ys;
  zz = (k - 0.5) * _dom.dz + _dom.zs;

  // perform tri-linear interpolation
  real dsdx = 0.5*(s[Ce] - s[Cw]) * ddx;
  real dsdy = 0.5*(s[Cn] - s[Cs]) * ddy;
  real dsdz = 0.5*(s[Ct] - s[Cb]) * ddz;
  ss[node + NNODES*part] = s[C] + dsdx*(x - xx) + dsdy*(y - yy) + dsdz*(z - zz)
    - s_parts[part].s;

  // set sswall equal to interfering wall s
  sswall = (parts[part].nodes[node] == WEST_WALL)  *bc_s->sWD
         + (parts[part].nodes[node] == EAST_WALL)  *bc_s->sED
         + (parts[part].nodes[node] == SOUTH_WALL) *bc_s->sSD
         + (parts[part].nodes[node] == NORTH_WALL) *bc_s->sND
         + (parts[part].nodes[node] == BOTTOM_WALL)*bc_s->sBD
         + (parts[part].nodes[node] == TOP_WALL)   *bc_s->sTD
         - s_parts[part].s;

  // set actual node value based on whether it is interfered with
  ss[node + NNODES*part] =
         (1 - oob) * ((parts[part].nodes[node] == -1) * ss[node + NNODES*part]
                    + (parts[part].nodes[node] <  -1) * sswall);
}

__global__ void scalar_lebedev_quadrature(part_struct *parts,
  part_struct_scalar *s_parts, int s_ncoeffs_max,
  real *ss, real *int_Ys_re, real *int_Ys_im)
{
  int part = blockIdx.x;
  int coeff = blockIdx.y;
  int node = threadIdx.x;

  if (coeff < s_parts[part].ncoeff) {
    /* Calculate integrand at each node */
    int j = part*NNODES*s_ncoeffs_max + coeff*NNODES + node;

    int n = _s_nn[coeff];
    int m = _s_mm[coeff];
    real theta = _node_t[node];
    real phi = _node_p[node];
    real N_nm = nnm(n, m);
    real P_nm = pnm(n, m, theta);

    // Precalculate things we use more than once
    real cmphi = cos(m * phi);
    real smphi = sin(m * phi);

    int stride = node + part*NNODES;

    int_Ys_re[j] = N_nm*P_nm*ss[stride]*cmphi;
    int_Ys_im[j] = -N_nm*P_nm*ss[stride]*smphi;

    __syncthreads();

    /* Compute partial sum of Lebedev quadrature (scalar product) */
    // put sum into first node position for each coeff for each particle
    if (node == 0) {
      int_Ys_re[j] *= _A1;
      int_Ys_im[j] *= _A1;
      for (int i = 1; i < 6; i++) {
        int_Ys_re[j] += _A1 * int_Ys_re[j+i];
        int_Ys_im[j] += _A1 * int_Ys_im[j+i];
      }
      for (int i = 6; i < 18; i++) {
        int_Ys_re[j] += _A2 * int_Ys_re[j+i];
        int_Ys_im[j] += _A2 * int_Ys_im[j+i];
      }
      for (int i = 18; i < 26; i++) {
        int_Ys_re[j] += _A3 * int_Ys_re[j+i];
        int_Ys_im[j] += _A3 * int_Ys_im[j+i];
      }
    }
  }
}

__global__ void pack_s_sums_e(real *sum_send_e, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    for (int ti = _bins.Gcc._ie; ti <= _bins.Gcc._ieb; ti++) {
      cbin = GFX_LOC(ti, tj, tk, s1b, s2b);
      c2b = tj + tk * s1b + (ti - _bins.Gcc._ie) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_e[sp0] = int_Ys_re[psum_ind];
          sum_send_e[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over ti planes
  }
}

__global__ void pack_s_sums_w(real *sum_send_w, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    for (int ti = _bins.Gcc._isb; ti <= _bins.Gcc._is; ti++) {
      cbin = GFX_LOC(ti, tj, tk, s1b, s2b);
      c2b = tj + tk * s1b + (ti - _bins.Gcc._isb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_w[sp0] = int_Ys_re[psum_ind];
          sum_send_w[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over ti
  }
}

__global__ void pack_s_sums_n(real *sum_send_n, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    for (int tj = _bins.Gcc._je; tj <= _bins.Gcc._jeb; tj++) {
      cbin = GFY_LOC(ti, tj, tk, s1b, s2b);
      c2b = tk + ti * s1b + (tj - _bins.Gcc._je) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_n[sp0] = int_Ys_re[psum_ind];
          sum_send_n[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over tj planes
  }
}

__global__ void pack_s_sums_s(real *sum_send_s, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    for (int tj = _bins.Gcc._jsb; tj <= _bins.Gcc._js; tj++) {
      cbin = GFY_LOC(ti, tj, tk, s1b, s2b);
      c2b = tk + ti * s1b + (tj - _bins.Gcc._jsb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_s[sp0] = int_Ys_re[psum_ind];
          sum_send_s[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over tj planes
  }
}

__global__ void pack_s_sums_t(real *sum_send_t, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    for (int tk = _bins.Gcc._ke; tk <= _bins.Gcc._keb; tk++) {
      cbin = GFZ_LOC(ti, tj, tk, s1b, s2b);
      c2b = ti + tj * s1b + (tk - _bins.Gcc._ke) * s2b;

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_t[sp0] = int_Ys_re[psum_ind];
          sum_send_t[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over tk planes
  }
}

__global__ void pack_s_sums_b(real *sum_send_b, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    for (int tk = _bins.Gcc._ksb; tk <= _bins.Gcc._ks; tk++) {
      cbin = GFZ_LOC(ti, tj, tk, s1b, s2b);
      c2b = ti + tj * s1b + (tk - _bins.Gcc._ksb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          sum_send_b[sp0] = int_Ys_re[psum_ind];
          sum_send_b[sp1] = int_Ys_im[psum_ind];
        }
      }
    } // loop over tk planes
  }
}

__global__ void unpack_s_sums_e(real *sum_recv_e, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    for (int ti = _bins.Gcc._ie; ti <= _bins.Gcc._ieb; ti++) {
      cbin = GFX_LOC(ti, tj, tk, s1b, s2b);
      c2b = tj + tk * s1b + (ti - _bins.Gcc._ie) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_e[sp0];
          int_Ys_im[psum_ind] += sum_recv_e[sp1];
        }
      }
    } // loop over ti
  }
}

__global__ void unpack_s_sums_w(real *sum_recv_w, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tj = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tk = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFX indices
  int s1b = _bins.Gcc.jnb;
  int s2b = s1b * _bins.Gcc.knb;

  if (tj < _bins.Gcc.jnb && tk < _bins.Gcc.knb) {
    for (int ti = _bins.Gcc._isb; ti <= _bins.Gcc._is; ti++) {
      cbin = GFX_LOC(ti, tj, tk, s1b, s2b);
      c2b = tj + tk * s1b + (ti - _bins.Gcc._isb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_w[sp0];
          int_Ys_im[psum_ind] += sum_recv_w[sp1];
        }
      }
    } // loop over ti
  }
}

__global__ void unpack_s_sums_n(real *sum_recv_n, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    for (int tj = _bins.Gcc._je; tj <= _bins.Gcc._jeb; tj++) {
      cbin = GFY_LOC(ti, tj, tk, s1b, s2b);
      c2b = tk + ti * s1b + (tj - _bins.Gcc._je) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_n[sp0];
          int_Ys_im[psum_ind] += sum_recv_n[sp1];
        }
      }
    } // loop over tj
  }
}

__global__ void unpack_s_sums_s(real *sum_recv_s, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int tk = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int ti = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFY indices
  int s1b = _bins.Gcc.knb;
  int s2b = s1b * _bins.Gcc.inb;

  if (tk < _bins.Gcc.knb && ti < _bins.Gcc.inb) {
    for (int tj = _bins.Gcc._jsb; tj <= _bins.Gcc._js; tj++) {
      cbin = GFY_LOC(ti, tj, tk, s1b, s2b);
      c2b = tk + ti * s1b + (tj - _bins.Gcc._jsb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_s[sp0];
          int_Ys_im[psum_ind] += sum_recv_s[sp1];
        }
      }
    } // loop over tj
  }
}

__global__ void unpack_s_sums_t(real *sum_recv_t, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    for (int tk = _bins.Gcc._ke; tk <= _bins.Gcc._keb; tk++) {
      cbin = GFZ_LOC(ti, tj, tk, s1b, s2b);
      c2b = ti + tj * s1b + (tk - _bins.Gcc._ke) * s2b;

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_t[sp0];
          int_Ys_im[psum_ind] += sum_recv_t[sp1];
        }
      }
    } // loop over tk
  }
}

__global__ void unpack_s_sums_b(real *sum_recv_b, int *offset, int *bin_start,
  int *bin_count, int *part_ind, int s_ncoeffs_max,
  real *int_Ys_re, real *int_Ys_im)
{
  int ti = blockIdx.x * blockDim.x + threadIdx.x; // bin index 
  int tj = blockIdx.y * blockDim.y + threadIdx.y;

  int cbin;       // bin index
  int c2b;        // bin index in 2-d plane
  int pp;         // particle index
  int dest;       // destination for particle partial sums in packed array
  int sp0, sp1;   // scalar product strides for (Ylm, s)
  int psum_ind;   // index of partial sum in each scalar product

  // Custom GFZ indices
  int s1b = _bins.Gcc.inb;
  int s2b = s1b * _bins.Gcc.jnb;

  if (ti < _bins.Gcc.inb && tj < _bins.Gcc.jnb) {
    for (int tk = _bins.Gcc._ksb; tk <= _bins.Gcc._ks; tk++) {
      cbin = GFZ_LOC(ti, tj, tk, s1b, s2b);
      c2b = ti + tj * s1b + (tk - _bins.Gcc._ksb) * s2b; // two planes

      // Loop through each bin's particles 
      // Each bin is offset by offset[cbin] (from excl. prefix scan)
      // Each particle is then offset from that
      for (int i = 0; i < bin_count[cbin]; i++) {
        pp = part_ind[bin_start[cbin] + i];
        dest = offset[c2b] + i;

        for (int coeff = 0; coeff < s_ncoeffs_max; coeff++) {
          // Packing: part varies slowest, coeff varies quickest, sp middle
          sp0 = coeff + s_ncoeffs_max*SP_YS_RE + s_ncoeffs_max*SNSP*dest;    // Ys_re
          sp1 = coeff + s_ncoeffs_max*SP_YS_IM + s_ncoeffs_max*SNSP*dest;    // Ys_im

          // Partial sums: part varies slowest, node quickest, coeff middle
          // Partial sums are stored in index for node = 0
          psum_ind = pp*NNODES*s_ncoeffs_max + coeff*NNODES;

          int_Ys_re[psum_ind] += sum_recv_b[sp0];
          int_Ys_im[psum_ind] += sum_recv_b[sp1];
        }
      }
    } // loop over tk
  }
}

__device__ real X_sn(int n, real theta, real phi,
  int pp, part_struct_scalar *s_parts)
{
  int coeff = 0;
  for(int j = 0; j < n; j++) coeff += 2*j+1;

  real sum = 0.;
  for(int m = -n; m <= n; m++) {
    sum += Nnm(n,m)*Pnm(n,m,theta)
      *(s_parts[pp].anm_re[coeff]*cos(m*phi)
      - s_parts[pp].anm_im[coeff]*sin(m*phi));
    coeff++;
  }
  return sum;
}

__global__ void scalar_part_BC(real *s, int *phase, int *phase_shell,
  part_struct *parts, part_struct_scalar *s_parts)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x + DOM_BUF;
  int tj = blockDim.y*blockIdx.y + threadIdx.y + DOM_BUF;
  int CC;
  real x, y, z;         // scalar node location Cartesian
  real Xp, Yp, Zp;      // particle position
  real r, theta, phi;   // velocity node location spherical
  real ss_tmp;          // temporary scalar
  int P, PS;            // phase, phase_shell
  real a;               // particle radius
  int order;            // particle scalar order
  real sp;              // particle scalar

  if (ti <= _dom.Gcc._ie && tj <= _dom.Gcc._je) {
    for (int k = _dom.Gcc._ks; k <= _dom.Gcc._ke; k++) {
      CC = GCC_LOC(ti, tj, k, _dom.Gcc.s1b, _dom.Gcc.s2b);

      // Position of current thread
      x = (ti-0.5) * _dom.dx + _dom.xs;
      y = (tj-0.5) * _dom.dy + _dom.ys;
      z = (k-0.5) * _dom.dz + _dom.zs;

      // get particle number and phase_shell number
      P = phase[CC];
      PS = phase_shell[CC];

      if(P > -1) {
        a = parts[P].r;
        Xp = parts[P].x;
        Yp = parts[P].y;
        Zp = parts[P].z;
        order = s_parts[P].order;
        sp = s_parts[P].s;
      } else {
        a = (_dom.dx + _dom.dy + _dom.dz) / 3.;
        Xp = (ti-0.5) * _dom.dx + _dom.xs + a;
        Yp = (tj-0.5) * _dom.dy + _dom.ys + a;
        Zp = (k-0.5) * _dom.dz + _dom.zs + a;
        // set order = -1, so it won't enter the "for loop"
        order = -1;
        sp = 0.;
      }

      // Position in particle frame
      x -= Xp;
      y -= Yp;
      z -= Zp;
      xyz2rtp(x, y, z, &r, &theta, &phi);

      // calculate analytic solution
      real ar = a / r;
      real ra = r / a;
      ss_tmp = sp;
      for(int n = 0; n <= order; n++) {
        ss_tmp += (pow(ra,n) - pow(ar,n+1)) * X_sn(n, theta, phi, P, s_parts);
      }

      // phase_shell = 1 means normal nodes, phase_shell = 0 means pressure nodes
      // phase shell cells: ss_tmp
      //       inner cells: sp, particle scalar
      //       fluid cells: s[CC]
      s[CC] = ss_tmp * (P > -1 && PS < 1)
            + sp     * (P > -1 && PS > 0)
            + s[CC]  * (P <= -1);
    }
  }
}

__global__ void scalar_compute_coeffs(part_struct *parts,
  part_struct_scalar *s_parts, int s_ncoeffs_max, int nparts,
  real *int_Ys_re, real *int_Ys_im)
{

  int coeff = threadIdx.x;
  int part = blockIdx.x;

  // precalculate constants
  real ars = parts[part].r / s_parts[part].rs;
  real rsa = s_parts[part].rs / parts[part].r;

  if (coeff < s_parts[part].ncoeff && part < nparts) {
    int j = part * NNODES * s_ncoeffs_max + coeff * NNODES + 0;
    int n = _s_nn[coeff];
    real A = pow(rsa, n) - pow(ars, n+1.);
    s_parts[part].anm_re[coeff] = int_Ys_re[j] / A;
    s_parts[part].anm_im[coeff] = int_Ys_im[j] / A;

    __syncthreads();

    // calculate heat flux for each particle
    if (coeff == 0) {
      s_parts[part].q = 2. * sqrt(PI) * parts[part].r * s_parts[part].anm_re[0];
    }
  }
}

__global__ void scalar_compute_error(real lamb_cut_scalar, int s_ncoeffs_max, int nparts,
  part_struct_scalar *s_parts, real *s_part_errors)
{
  int part = blockIdx.x;
  int coeff = threadIdx.x;

  real div = 0.;
  real max = DBL_MIN;

  __shared__ real s_coeffs[S_MAX_COEFFS * SNSP];
  __shared__ real s_coeffs0[S_MAX_COEFFS * SNSP];
  __shared__ real s_max[S_MAX_COEFFS];

  if (part < nparts && coeff < s_ncoeffs_max) {

    s_coeffs[coeff + s_ncoeffs_max * 0] = s_parts[part].anm_re[coeff];
    s_coeffs[coeff + s_ncoeffs_max * 1] = s_parts[part].anm_im[coeff];

    s_coeffs0[coeff + s_ncoeffs_max * 0] = s_parts[part].anm_re0[coeff];
    s_coeffs0[coeff + s_ncoeffs_max * 1] = s_parts[part].anm_im0[coeff];

    s_max[coeff] = DBL_MIN;

    __syncthreads();
    
    // If coefficient has a large enough magnitude (relative to 0th order coeff)
    //  calculate the error
    for (int i = 0; i < SNSP; i++) {
      int c = coeff + s_ncoeffs_max * i;

      // Determine if current coefficient has large enough value compared to 0th
      // (also, make sure it's large enough so we don't get issues with close-to-zero
      //  errors)
      // (also, if zeroth order is 0, ignore)
      real curr_val = s_coeffs[c];
      real zeroth_val = s_coeffs[0 + s_ncoeffs_max * i];
      int flag = (fabs(curr_val) > fabs(lamb_cut_scalar*zeroth_val)) *
                  (fabs(curr_val) > 1.e-16) *
                  (fabs(zeroth_val) > DBL_MIN);

      // If flag == 1, set scoeff equal to error value
      // If flag == 0, set scoeff equal to zero (no error)
      div = fabs(curr_val);
      div += (1.e-16 - div) * (div < 1.e-16);
      real curr_val0 = s_coeffs0[c];

      s_coeffs[c] = (real) flag * fabs(curr_val - curr_val0) / div;

      // See if current error is the max we've seen so far over all the
      // coefficients of a given order, set if so
      s_max[coeff] += (s_coeffs[c] - s_max[coeff]) * (s_coeffs[c] > s_max[coeff]);
    }

    __syncthreads();

    // We've now calculated the error for each "large enough" coefficients and
    //  found the maximum over all coefficients of a given order. Now, each
    //  order has a maximum, and we need to find the max over these
    if (coeff == 0) {
      for (int i = 0; i < s_ncoeffs_max; i++) {
        max += (s_max[i] - max) * (s_max[i] > max);  
      }
      s_part_errors[part] = max;
    }
  }
}

__global__ void scalar_store_coeffs(part_struct_scalar *s_parts, int nparts,
  int s_ncoeffs_max)
{
  int part = blockIdx.x;
  int coeff = threadIdx.x;
  if (part < nparts && coeff < s_ncoeffs_max) {
   s_parts[part].anm_re0[coeff] = s_parts[part].anm_re[coeff];
   s_parts[part].anm_im0[coeff] = s_parts[part].anm_im[coeff];
  }
}

__global__ void update_part_scalar(part_struct *parts,
  part_struct_scalar *s_parts, real time, real dt, real s_k)
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x; // particle index

  real vol = 4./3. * PI * parts[pp].r*parts[pp].r*parts[pp].r;
  real m = vol * parts[pp].rho;
  // prepare s for next timestep
  s_parts[pp].s += (float)s_parts[pp].update * s_parts[pp].q * s_k * dt / m /s_parts[pp].cp;
}

__global__ void scalar_part_fill(real *s, int *phase,
  part_struct_scalar *s_parts)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x + DOM_BUF;
  int tj = blockDim.y*blockIdx.y + threadIdx.y + DOM_BUF;
  int CC, P;
  real sp = 0.;

  if (ti <= _dom.Gcc._ie && tj <= _dom.Gcc._je) {
    for (int k = _dom.Gcc._ks; k <= _dom.Gcc._ke; k++) {
      CC = GCC_LOC(ti, tj, k, _dom.Gcc.s1b, _dom.Gcc.s2b);
      P = phase[CC];
      if(P > -1) sp = s_parts[P].s;
      s[CC] = sp * (P > -1) + s[CC] * (P == -1);
    }
  }
}
